#include "hip/hip_runtime.h"
#define CATCH_CONFIG_ENABLE_BENCHMARKING

#include "../extern/Catch2/single_include/catch2/catch.hpp"
#include "../include/device_matrix.h"
#include "../include/nearest_neighbor.h"
#include "../include/utils.h"
#include "utils.h"

#include <vector>

TEST_CASE("Nearest neighbor labeling", "[nearest_neighbour]")
{
	GIVEN("A distance matrix")
	{
		const auto vec = std::vector<double>{
			1.0, 4.0, 3.0, 2.0, //
			2.0, 1.1, 4.1, 3.0, //
			3.0, 2.0, 1.2, 4.2, //
			4.3, 3.0, 2.0, 1.3, //
		};
		const auto rows = 4_rows;
		const auto cols = 4_cols;
		const auto col_maj = true;

		const auto solution_min = std::vector<size_t>{0, 1, 2, 3};
		const auto solution_max = std::vector<size_t>{3, 0, 1, 2};

		const auto solution_dist_min = std::vector<double>{1.0, 1.1, 1.2, 1.3};
		const auto solution_dist_max = std::vector<double>{4.3, 4.0, 4.1, 4.2};

		const auto size_nn = Size{2_rows, 2_cols};

		const auto mat = device_matrix<double>{vec, rows, cols, col_maj};

		WHEN("the nearest neighbors are computed")
		{
			constexpr auto min = true;
			const auto neigh_dist_min = nearest_neighbor_w_distances(mat, size_nn, min);
			const auto neigh_dist_max = nearest_neighbor_w_distances(mat, size_nn, !min);

			const auto neigh_min = std::get<0>(neigh_dist_min);
			const auto dist_min = std::get<1>(neigh_dist_min);
			const auto neigh_max = std::get<0>(neigh_dist_max);
			const auto dist_max = std::get<1>(neigh_dist_max);

			THEN("they equal the solution")
			{
				CHECK(neigh_min.size() == size_nn);
				CHECK(neigh_max.size() == size_nn);

				CHECK(get_host_vector(neigh_min) == solution_min);
				CHECK(get_host_vector(neigh_max) == solution_max);

				CHECK(dist_min.size() == size_nn);
				CHECK(dist_max.size() == size_nn);

				CHECK(get_host_vector(dist_min) == solution_dist_min);
				CHECK(get_host_vector(dist_max) == solution_dist_max);
			}
		}
	}
}

TEST_CASE("Nearest neighbor labeling (benchmark)", "[nearest_neighbor][!benchmark]")
{
	const auto rows = Rows{PATCH_COUNT_BENCHMARK.value()};
	const auto cols = Cols{LABEL_COUNT_BENCHMARK.value()};
	const auto matrix = rand_matrix(Size{rows, cols}, double{});
	constexpr auto min = true;

	BENCHMARK("Nearest neighbor")
	{
		return nearest_neighbor(matrix, PATCH_COUNT_SIZE_BENCHMARK, min);
	};
}