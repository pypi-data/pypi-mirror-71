#include "hip/hip_runtime.h"
#define CATCH_CONFIG_ENABLE_BENCHMARKING

#include "../extern/Catch2/single_include/catch2/catch.hpp"
#include "../include/device_matrix.h"
#include "../include/extract_patches.h"
#include "../include/log_polar_transform.h"
#include "../include/utils.h"
#include "data_similar.h"
#include "utils.h"

#include <cstddef>

TEMPLATE_TEST_CASE("Log-polar transform (w/o embedding)", "[log_polar_transform][template]", float,
				   double, vec3<float>, vec3<double>)
{
	GIVEN("Patches (embedded in a black background)")
	{
		const auto patches_col_maj = data_similar::patches_rhs(TestType{});
		const auto patches_row_maj = to_row_maj(patches_col_maj);
		const auto solution = data_similar::patches_rhs_log_polar(TestType{});
		const auto solution_t = transpose(solution);

		const auto lp_rows = Rows{solution.rows_patches()};
		const auto lp_cols = Cols{solution.cols_patches()};
		const auto already_embedded = false;
		const auto transposed = true;

		WHEN("applying log-polar transformation")
		{
			const auto log_polar_col_maj = log_polar_transform(patches_col_maj, lp_rows, lp_cols,
															   already_embedded, !transposed);
			const auto log_polar_col_maj_t = log_polar_transform(patches_col_maj, lp_rows, lp_cols,
																 already_embedded, transposed);

			const auto log_polar_row_maj = log_polar_transform(patches_row_maj, lp_rows, lp_cols,
															   already_embedded, !transposed);
			const auto log_polar_row_maj_t = log_polar_transform(patches_row_maj, lp_rows, lp_cols,
																 already_embedded, transposed);

			THEN("the interpolation equals the known solution")
			{
				const auto eps = 0.0;
				const auto margin = 0.01;
				is_equal(log_polar_col_maj, solution, eps, margin);
				is_equal(log_polar_col_maj_t, solution_t, eps, margin);
				is_equal(log_polar_row_maj, to_row_maj(solution), eps, margin);
				is_equal(log_polar_row_maj_t, to_row_maj(solution_t), eps, margin);
			}
		}
	}
}

TEMPLATE_TEST_CASE("Log-polar inverse transform (w/o embedding)", "[log_polar_transform][template]",
				   float, double, vec3<float>, vec3<double>)
{
	GIVEN("Log polar transformed patches (embedded in a black background)")
	{
		const auto already_embedded = false;
		const auto transposed = true;

		const auto original = data_similar::patches_rhs(TestType{});
		const auto patches_col_maj =
			log_polar_transform(original, 80_rows, 100_cols, already_embedded, !transposed);
		const auto patches_row_maj = to_row_maj(patches_col_maj);
		const auto patches_col_maj_t = transpose(patches_col_maj);
		const auto patches_row_maj_t = transpose(patches_row_maj);

		const auto solution = data_similar::patches_rhs(TestType{});

		const auto rows = Rows{solution.rows_patches()};
		const auto cols = Cols{solution.cols_patches()};

		WHEN("applying inverse log-polar transformation")
		{
			const auto log_polar_inv_col_maj =
				log_polar_inv_transform(patches_col_maj, rows, cols, already_embedded, !transposed);
			const auto log_polar_inv_col_maj_t = log_polar_inv_transform(
				patches_col_maj_t, rows, cols, already_embedded, transposed);

			const auto log_polar_inv_row_maj =
				log_polar_inv_transform(patches_row_maj, rows, cols, already_embedded, !transposed);
			const auto log_polar_inv_row_maj_t = log_polar_inv_transform(
				patches_row_maj_t, rows, cols, already_embedded, transposed);

			THEN("the result equals the original patches")
			{
				const auto eps = 0.0;
				const auto margin = 0.01;
				is_equal(log_polar_inv_col_maj, solution, eps, margin);
				is_equal(log_polar_inv_col_maj_t, solution, eps, margin);
				is_equal(log_polar_inv_row_maj, to_row_maj(solution), eps, margin);
				is_equal(log_polar_inv_row_maj_t, to_row_maj(solution), eps, margin);
			}
		}
	}
}

// TODO: Maybe add a test with embedded log-polar transform. Though, the function implementation is
// the same as w/o embedding, just with a different interpolation radius.

TEST_CASE("Log polar transform (benchmark)", "[log_polar_transform_benchmark][!benchmark]")
{
	const auto patches = rand_patches(PATCH_COUNT_BENCHMARK, PATCH_SIZE_BENCHMARK, BENCHMARK_T{});
	const auto embed = true;
	const auto transpose = true;

	BENCHMARK("Log-polar transform")
	{
		return log_polar_transform(patches, 16_rows, 16_cols, embed, transpose);
	};
}