#include "../include/device_matrix.h"
#include "../include/device_patches.h"
#include "../include/utils.h"
#include "../include/vec3.h"

#include <hip/hip_complex.h>

#include <vector>

namespace data_afmt
{
template<typename Float>
auto patch0(Float /*tag*/)
{
	return std::vector<Float>{
		1.0, 0.0, 0.0, 0.0, // col 0
		0.0, 0.0, 0.0, 0.0, // col 1
		0.0, 0.0, 0.0, 0.0, // col 2
		0.0, 0.0, 0.0, 0.0, // col 3
	};
}
template<typename Float>
auto patch1(Float /*tag*/)
{
	return std::vector<Float>{
		1.0, 1.0, 0.0, 0.0, // col 0
		1.0, 1.0, 0.0, 0.0, // col 1
		0.0, 0.0, 0.0, 0.0, // col 2
		0.0, 0.0, 0.0, 0.0, // col 3
	};
}
template<typename Float>
auto patch2(Float /*tag*/)
{
	return std::vector<Float>{
		0.0, 0.0, 0.0, 0.0, // col 0
		0.0, 1.0, 1.0, 0.0, // col 1
		0.0, 1.0, 1.0, 0.0, // col 2
		0.0, 0.0, 0.0, 0.0, // col 3
	};
}

template<typename Complex>
auto patch_afmt0(Complex /*tag*/)
{
	return std::vector<Complex>{
		Complex{1.571f, 0.0f}, Complex{1.571f, 0.0f}, Complex{1.571f, 0.0f}, // col 0
		Complex{1.571f, 0.0f}, Complex{1.571f, 0.0f}, Complex{1.571f, 0.0f}, // col 1
		Complex{1.571f, 0.0f}, Complex{1.571f, 0.0f}, Complex{1.571f, 0.0f}, // col 2
		Complex{1.571f, 0.0f}, Complex{1.571f, 0.0f}, Complex{1.571f, 0.0f}, // col 3
	};
}
template<typename Complex>
auto patch_afmt1(Complex /*tag*/)
{
	return std::vector<Complex>{
		Complex{0.0f, 0.0f}, Complex{-1.874f, -1.874f}, Complex{-3.749f, 0.0f}, // col 0
		Complex{0.0f, 0.0f}, Complex{-1.874f, 5.016f},	Complex{3.142f, 6.890f}, // col 1
		Complex{0.0f, 0.0f}, Complex{5.016f, 5.016f},	Complex{10.032f, 0.0f}, // col 2
		Complex{0.0f, 0.0f}, Complex{5.016f, -1.874f},	Complex{3.142f, -6.890f}, // col 3
	};
}
template<typename Complex>
auto patch_afmt2(Complex /*tag*/)
{
	return std::vector<Complex>{
		Complex{0.0f, 0.0f}, Complex{-4.111f, 4.111f},	 Complex{8.222f, 0.0f}, // col 0
		Complex{0.0f, 0.0f}, Complex{4.111f, -11.001f},	 Complex{-15.113f, 6.890f}, // col 1
		Complex{0.0f, 0.0f}, Complex{-11.001f, 11.001f}, Complex{22.003f, 0.0f}, // col 2
		Complex{0.0f, 0.0f}, Complex{11.001f, -4.111f},	 Complex{-15.113f, -6.890f}, // col 3
	};
}

template<typename Float, typename = std::enable_if_t<std::is_floating_point<Float>::value>>
device_patches<Float> patches_impl(Float tag)
{
	const auto patch0_ = patch0(tag);
	const auto patch1_ = patch1(tag);
	const auto patch2_ = patch2(tag);
	auto data = patch0_;
	data.insert(data.end(), patch1_.cbegin(), patch1_.cend());
	data.insert(data.end(), patch2_.cbegin(), patch2_.cend());

	return device_patches<Float>{data, 3_patches, 4_rows, 4_cols};
}

device_patches<float> patches(float tag)
{
	return patches_impl(tag);
}

device_patches<double> patches(double tag)
{
	return patches_impl(tag);
}

template<typename Complex>
device_patches<Complex> patches_afmt_impl(Complex tag)
{
	const auto patch0_ = patch_afmt0(tag);
	const auto patch1_ = patch_afmt1(tag);
	const auto patch2_ = patch_afmt2(tag);
	auto data = patch0_;
	data.insert(data.end(), patch1_.cbegin(), patch1_.cend());
	data.insert(data.end(), patch2_.cbegin(), patch2_.cend());

	return device_patches<Complex>{data, 3_patches, 3_rows, 4_cols};
}

device_patches<hipComplex> patches_afmt(hipComplex tag)
{
	return patches_afmt_impl(tag);
}

device_patches<hipDoubleComplex> patches_afmt(hipDoubleComplex tag)
{
	return patches_afmt_impl(tag);
}
} // namespace data_afmt