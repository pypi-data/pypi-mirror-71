#include "hip/hip_runtime.h"
#include "../extern/Catch2/single_include/catch2/catch.hpp"
#include "../include/device_matrix.h"
#include "../include/device_patches.h"
#include "../include/utils.h"
#include "../include/vec3.h"
#include "data_similar.h"

#include <vector>

namespace data_similar
{
/*
 ** Patches which are common to multiple functions.
 */
// "rotation by 0deg"
template<typename Float>
auto patch0(Float /*tag*/)
{
	return std::vector<Float>{
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 0
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 1
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 2
		0.0, 0.0, 1.0, 1.0, 0.0, 0.0, 0.0, 0.0, // col 3
		0.0, 0.0, 1.0, 1.0, 0.0, 0.0, 0.0, 0.0, // col 4
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 5
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 6
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 7
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 8
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 9
	};
}
// rotation by -90deg
template<typename Float>
auto patch90(Float /*tag*/)
{
	return std::vector<Float>{
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 0
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 1
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 2
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 3
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 4
		0.0, 0.0, 1.0, 1.0, 0.0, 0.0, 0.0, 0.0, // col 5
		0.0, 0.0, 1.0, 1.0, 0.0, 0.0, 0.0, 0.0, // col 6
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 7
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 8
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 9
	};
}
// rotation by -180deg
template<typename Float>
auto patch180(Float /*tag*/)
{
	return std::vector<Float>{
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 0
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 1
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 2
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 3
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 4
		0.0, 0.0, 0.0, 0.0, 1.0, 1.0, 0.0, 0.0, // col 5
		0.0, 0.0, 0.0, 0.0, 1.0, 1.0, 0.0, 0.0, // col 6
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 7
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 8
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 9
	};
};
// rotation by -180deg
template<typename Float>
auto patch270(Float /*tag*/)
{
	return std::vector<Float>{
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 0
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 1
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 2
		0.0, 0.0, 0.0, 0.0, 1.0, 1.0, 0.0, 0.0, // col 3
		0.0, 0.0, 0.0, 0.0, 1.0, 1.0, 0.0, 0.0, // col 4
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 5
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 6
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 7
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 8
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 9
	};
}

// different patch "rotated by 0deg"
template<typename Float>
auto patch_0(Float /*tag*/)
{
	return std::vector<Float>{
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 0
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 1
		0.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 0.0, // col 2
		0.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 0.0, // col 3
		0.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 0.0, // col 4
		0.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 0.0, // col 5
		0.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 0.0, // col 6
		0.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 0.0, // col 7
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 8
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 9
	};
}
// scaled by 2/3
template<typename Float>
auto patch_x2(Float /*tag*/)
{
	return std::vector<Float>{
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 0
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 1
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 2
		0.0, 0.0, 1.0, 1.0, 1.0, 1.0, 0.0, 0.0, // col 3
		0.0, 0.0, 1.0, 1.0, 1.0, 1.0, 0.0, 0.0, // col 4
		0.0, 0.0, 1.0, 1.0, 1.0, 1.0, 0.0, 0.0, // col 5
		0.0, 0.0, 1.0, 1.0, 1.0, 1.0, 0.0, 0.0, // col 6
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 7
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 8
		0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, // col 9
	};
}

// log polar transformation of patch0
template<typename Float>
auto patch0_lp(Float /*tag*/)
{
	return std::vector<Float>{
		0.245f, 0.240f, 0.228f, 0.201f, 0.145f, 0.040f, -0.037f, 0.000f, // col 0
		0.243f, 0.236f, 0.219f, 0.185f, 0.122f, 0.034f, 0.001f,	 0.000f, // col 1
		0.245f, 0.240f, 0.228f, 0.201f, 0.145f, 0.040f, -0.037f, 0.000f, // col 2
		0.250f, 0.250f, 0.249f, 0.245f, 0.227f, 0.150f, -0.038f, -0.019f, // col 3
		0.255f, 0.260f, 0.272f, 0.299f, 0.355f, 0.460f, 0.555f,	 0.357f, // col 4
		0.257f, 0.265f, 0.283f, 0.324f, 0.424f, 0.666f, 1.082f,	 1.173f, // col 5
		0.255f, 0.260f, 0.272f, 0.299f, 0.355f, 0.460f, 0.555f,	 0.357f, // col 6
		0.250f, 0.250f, 0.249f, 0.245f, 0.227f, 0.150f, -0.038f, -0.019f, // col 3
	};
}

// log polar transformation of patch90
// Rotation in the patch results in translation in the log polar transformation.
template<typename Float>
auto patch90_lp(Float /*tag*/)
{
	return std::vector<Float>{
		0.255f, 0.260f, 0.272f, 0.299f, 0.355f, 0.460f, 0.555f,	 0.357f, // col 0
		0.250f, 0.250f, 0.249f, 0.245f, 0.227f, 0.150f, -0.038f, -0.019f, // col 1
		0.245f, 0.240f, 0.228f, 0.201f, 0.145f, 0.040f, -0.037f, 0.000f, // col 2
		0.243f, 0.236f, 0.219f, 0.185f, 0.122f, 0.034f, 0.001f,	 0.000f, // col 3
		0.245f, 0.240f, 0.228f, 0.201f, 0.145f, 0.040f, -0.037f, 0.000f, // col 4
		0.250f, 0.250f, 0.249f, 0.245f, 0.227f, 0.150f, -0.038f, -0.019f, // col 5
		0.255f, 0.260f, 0.272f, 0.299f, 0.355f, 0.460f, 0.555f,	 0.357f, // col 6
		0.257f, 0.265f, 0.283f, 0.324f, 0.424f, 0.666f, 1.082f,	 1.173f, // col 7
	};
}

// log polar transformation of patch180
// Rotation in the patch results in translation in the log polar transformation.
template<typename Float>
auto patch180_lp(Float /*tag*/)
{
	return std::vector<Float>{
		0.255f, 0.260f, 0.272f, 0.299f, 0.355f, 0.460f, 0.555f,	 0.357f, // col 0
		0.257f, 0.265f, 0.283f, 0.324f, 0.424f, 0.666f, 1.082f,	 1.173f, // col 1
		0.255f, 0.260f, 0.272f, 0.299f, 0.355f, 0.460f, 0.555f,	 0.357f, // col 2
		0.250f, 0.250f, 0.249f, 0.245f, 0.227f, 0.150f, -0.038f, -0.019f, // col 3
		0.245f, 0.240f, 0.228f, 0.201f, 0.145f, 0.040f, -0.037f, 0.000f, // col 4
		0.243f, 0.236f, 0.219f, 0.185f, 0.122f, 0.034f, 0.001f,	 0.000f, // col 5
		0.245f, 0.240f, 0.228f, 0.201f, 0.145f, 0.040f, -0.037f, 0.000f, // col 6
		0.250f, 0.250f, 0.249f, 0.245f, 0.227f, 0.150f, -0.038f, -0.019f, // col 7
	};
}

// log polar transformation of patch270
// Rotation in the patch results in translation in the log polar transformation.
template<typename Float>
auto patch270_lp(Float /*tag*/)
{
	return std::vector<Float>{
		0.245f, 0.240f, 0.228f, 0.201f, 0.145f, 0.040f, -0.037f, 0.000f, // col 0
		0.250f, 0.250f, 0.249f, 0.245f, 0.227f, 0.150f, -0.038f, -0.019f, // col 1
		0.255f, 0.260f, 0.272f, 0.299f, 0.355f, 0.460f, 0.555f,	 0.357f, // col 2
		0.257f, 0.265f, 0.283f, 0.324f, 0.424f, 0.666f, 1.082f,	 1.173f, // col 3
		0.255f, 0.260f, 0.272f, 0.299f, 0.355f, 0.460f, 0.555f,	 0.357f, // col 4
		0.250f, 0.250f, 0.249f, 0.245f, 0.227f, 0.150f, -0.038f, -0.019f, // col 5
		0.245f, 0.240f, 0.228f, 0.201f, 0.145f, 0.040f, -0.037f, 0.000f, // col 6
		0.243f, 0.236f, 0.219f, 0.185f, 0.122f, 0.034f, 0.001f,	 0.000f, // col 7
	};
}

// log polar transformation of patch_x2
template<typename Float>
auto patch_x2_lp(Float /*tag*/)
{
	return std::vector<Float>{
		1.000f, 1.000f, 1.000f, 1.000f, 1.000f, 1.000f, 1.037f, 0.715f, // col 0
		1.000f, 1.000f, 1.000f, 1.000f, 1.000f, 1.000f, 1.007f, 1.135f, // col 1
		1.000f, 1.000f, 1.000f, 1.000f, 1.000f, 1.000f, 1.037f, 0.715f, // col 2
		1.000f, 1.000f, 1.000f, 1.000f, 1.000f, 1.000f, 1.007f, 1.135f, // col 3
		1.000f, 1.000f, 1.000f, 1.000f, 1.000f, 1.000f, 1.037f, 0.715f, // col 4
		1.000f, 1.000f, 1.000f, 1.000f, 1.000f, 1.000f, 1.007f, 1.135f, // col 5
		1.000f, 1.000f, 1.000f, 1.000f, 1.000f, 1.000f, 1.037f, 0.715f, // col 6
		1.000f, 1.000f, 1.000f, 1.000f, 1.000f, 1.000f, 1.007f, 1.135f, // col 7
	};
}

template<typename Float, typename = std::enable_if_t<std::is_floating_point<Float>::value>>
device_patches<Float> patches_lhs_impl(Float tag)
{
	const auto p = patch0(tag);
	const auto p_ = patch_0(tag);
	auto data = p;
	data.insert(data.end(), p.cbegin(), p.cend());
	data.insert(data.end(), p_.cbegin(), p_.cend());
	data.insert(data.end(), p_.cbegin(), p_.cend());

	return device_patches<Float>{data, 4_patches, 8_rows, 10_cols};
}

device_patches<float> patches_lhs(float tag)
{
	return patches_lhs_impl(tag);
}

device_patches<double> patches_lhs(double tag)
{
	return patches_lhs_impl(tag);
}

template<typename Float, typename = std::enable_if_t<std::is_floating_point<Float>::value>>
device_patches<Float> patches_rhs_impl(Float tag)
{
	const auto p0 = patch0(tag);
	const auto p90 = patch90(tag);
	const auto p_x2 = patch_x2(tag);
	const auto p00 = std::vector<Float>(p0.size(), 0.0);

	auto data = p0;
	data.insert(data.end(), p90.cbegin(), p90.cend());
	data.insert(data.end(), p_x2.cbegin(), p_x2.cend());
	data.insert(data.end(), p00.cbegin(), p00.cend());

	return device_patches<Float>{data, 4_patches, 8_rows, 10_cols};
}

device_patches<float> patches_rhs(float tag)
{
	return patches_rhs_impl(tag);
}

device_patches<double> patches_rhs(double tag)
{
	return patches_rhs_impl(tag);
}

template<typename Float>
device_patches<vec3<Float>> patches_lhs_impl(vec3<Float> /*tag*/)
{
	const auto p0 = patch0(Float{});
	const auto p90 = patch90(Float{});
	const auto p180 = patch180(Float{});
	const auto p_0 = patch_0(Float{});

	const auto ch0 = [&p0, &p_0] {
		auto tmp = p0;
		tmp.insert(tmp.end(), p0.cbegin(), p0.cend());
		tmp.insert(tmp.end(), p_0.cbegin(), p_0.cend());
		tmp.insert(tmp.end(), p_0.cbegin(), p_0.cend());
		return tmp;
	}();
	const auto ch1 = [&p90, &p_0] {
		auto tmp = p90;
		tmp.insert(tmp.end(), p90.cbegin(), p90.cend());
		tmp.insert(tmp.end(), p_0.cbegin(), p_0.cend());
		tmp.insert(tmp.end(), p_0.cbegin(), p_0.cend());
		return tmp;
	}();
	const auto ch2 = [&p180, &p_0] {
		auto tmp = p180;
		tmp.insert(tmp.end(), p180.cbegin(), p180.cend());
		tmp.insert(tmp.end(), p_0.cbegin(), p_0.cend());
		tmp.insert(tmp.end(), p_0.cbegin(), p_0.cend());
		return tmp;
	}();
	const auto d_ch0 = device_patches<Float>{ch0, 4_patches, 8_rows, 10_cols};
	const auto d_ch1 = device_patches<Float>{ch1, 4_patches, 8_rows, 10_cols};
	const auto d_ch2 = device_patches<Float>{ch2, 4_patches, 8_rows, 10_cols};

	return join(d_ch0, d_ch1, d_ch2);
}

device_patches<vec3<float>> patches_lhs(vec3<float> tag)
{
	return patches_lhs_impl(tag);
}

device_patches<vec3<double>> patches_lhs(vec3<double> tag)
{
	return patches_lhs_impl(tag);
}

template<typename Float>
device_patches<vec3<Float>> patches_rhs_impl(vec3<Float> /*tag*/)
{
	const auto p0 = patch0(Float{});
	const auto p90 = patch90(Float{});
	const auto p180 = patch180(Float{});
	const auto p270 = patch270(Float{});
	const auto p_x2 = patch_x2(Float{});
	const auto p00 = std::vector<Float>(p0.size(), 0.0);

	// The first patch consists of an image rotated by 0, 90, 180 degrees.
	// The second patch consists of an image rotated by 90, 180, 270 degrees and scaled by 2.
	// Hence both rgb patches are related by a 90 degree rotation and scale 2.
	// The third patch is scaled by 2.
	// The fourth patch is zero and acts as a control.
	const auto ch0 = [&p0, &p90, &p_x2, &p00] {
		auto tmp = p0;
		tmp.insert(tmp.end(), p90.cbegin(), p90.cend());
		tmp.insert(tmp.end(), p_x2.cbegin(), p_x2.cend());
		tmp.insert(tmp.end(), p00.cbegin(), p00.cend());
		return tmp;
	}();
	const auto ch1 = [&p90, &p180, &p_x2, &p00] {
		auto tmp = p90;
		tmp.insert(tmp.end(), p180.cbegin(), p180.cend());
		tmp.insert(tmp.end(), p_x2.cbegin(), p_x2.cend());
		tmp.insert(tmp.end(), p00.cbegin(), p00.cend());
		return tmp;
	}();
	const auto ch2 = [&p180, &p270, &p_x2, &p00] {
		auto tmp = p180;
		tmp.insert(tmp.end(), p270.cbegin(), p270.cend());
		tmp.insert(tmp.end(), p_x2.cbegin(), p_x2.cend());
		tmp.insert(tmp.end(), p00.cbegin(), p00.cend());
		return tmp;
	}();
	const auto d_ch0 = device_patches<Float>{ch0, 4_patches, 8_rows, 10_cols};
	const auto d_ch1 = device_patches<Float>{ch1, 4_patches, 8_rows, 10_cols};
	const auto d_ch2 = device_patches<Float>{ch2, 4_patches, 8_rows, 10_cols};

	return join(d_ch0, d_ch1, d_ch2);
}

device_patches<vec3<float>> patches_rhs(vec3<float> tag)
{
	return patches_rhs_impl(tag);
}

device_patches<vec3<double>> patches_rhs(vec3<double> tag)
{
	return patches_rhs_impl(tag);
}

template<typename Float, typename = std::enable_if_t<std::is_floating_point<Float>::value>>
device_patches<Float> patches_rhs_log_polar_impl(Float tag)
{
	const auto p0 = patch0_lp(tag);
	const auto p90 = patch90_lp(tag);
	const auto p_x2 = patch_x2_lp(tag);
	const auto p00 = std::vector<Float>(p0.size(), 0.0);

	auto data = p0;
	data.insert(data.end(), p90.cbegin(), p90.cend());
	data.insert(data.end(), p_x2.cbegin(), p_x2.cend());
	data.insert(data.end(), p00.cbegin(), p00.cend());

	return device_patches<Float>{data, 4_patches, 8_rows, 8_cols};
}

device_patches<float> patches_rhs_log_polar(float tag)
{
	return patches_rhs_log_polar_impl(tag);
}

device_patches<double> patches_rhs_log_polar(double tag)
{
	return patches_rhs_log_polar_impl(tag);
}

template<typename Float>
device_patches<vec3<Float>> patches_rhs_log_polar_impl(vec3<Float> /*tag*/)
{
	const auto p0 = patch0_lp(Float{});
	const auto p90 = patch90_lp(Float{});
	const auto p180 = patch180_lp(Float{});
	const auto p270 = patch270_lp(Float{});
	const auto p_x2 = patch_x2_lp(Float{});
	const auto p00 = std::vector<Float>(p0.size(), 0.0);

	// The first patch consists of an image rotated by 0, 90, 180 degrees.
	// The second patch consists of an image rotated by 90, 180, 270 degrees and scaled by 2.
	// Hence both rgb patches are related by a 90 degree rotation and scale 2.
	// The third patch is scaled by 2.
	// The fourth patch is zero and acts as a control.
	const auto ch0 = [&p0, &p90, &p_x2, &p00] {
		auto tmp = p0;
		tmp.insert(tmp.end(), p90.cbegin(), p90.cend());
		tmp.insert(tmp.end(), p_x2.cbegin(), p_x2.cend());
		tmp.insert(tmp.end(), p00.cbegin(), p00.cend());
		return tmp;
	}();
	const auto ch1 = [&p90, &p180, &p_x2, &p00] {
		auto tmp = p90;
		tmp.insert(tmp.end(), p180.cbegin(), p180.cend());
		tmp.insert(tmp.end(), p_x2.cbegin(), p_x2.cend());
		tmp.insert(tmp.end(), p00.cbegin(), p00.cend());
		return tmp;
	}();
	const auto ch2 = [&p180, &p270, &p_x2, &p00] {
		auto tmp = p180;
		tmp.insert(tmp.end(), p270.cbegin(), p270.cend());
		tmp.insert(tmp.end(), p_x2.cbegin(), p_x2.cend());
		tmp.insert(tmp.end(), p00.cbegin(), p00.cend());
		return tmp;
	}();
	const auto d_ch0 = device_patches<Float>{ch0, 4_patches, 8_rows, 8_cols};
	const auto d_ch1 = device_patches<Float>{ch1, 4_patches, 8_rows, 8_cols};
	const auto d_ch2 = device_patches<Float>{ch2, 4_patches, 8_rows, 8_cols};

	return join(d_ch0, d_ch1, d_ch2);
}

device_patches<vec3<float>> patches_rhs_log_polar(vec3<float> tag)
{
	return patches_rhs_log_polar_impl(tag);
}

device_patches<vec3<double>> patches_rhs_log_polar(vec3<double> tag)
{
	return patches_rhs_log_polar_impl(tag);
}

//! Perspective transformations between the test patches.
device_patches<float> perspective_matrices()
{
	const auto row_maj = false;
	const auto data = std::vector<float>{
		// projective matrix
		1.0, 0.0,  0.0,	  0.0, 1.0, 0.0,   0.0, 0.0, 1.0, // identity
		0.0, -1.0, 9.0,	  1.0, 0.0, 0.0,   0.0, 0.0, 1.0, // rotation by -90deg
		1.5, 0.0,  -2.25, 0.0, 1.5, -1.75, 0.0, 0.0, 1.0, // scaling by 2/3
		1.0, 0.0,  0.0,	  0.0, 1.0, 0.0,   0.0, 0.0, 1.0, // identity
	};

	return device_patches<float>{data, 4_patches, 3_rows, 3_cols, row_maj};
}

device_patches<float> clustering()
{
	const auto p0 = patch0(float{});
	const auto p90 = patch90(float{});
	const auto p180 = patch180(float{});
	const auto p270 = patch270(float{});
	const auto prand = std::vector<float>{
		// random patch
		0.5f, 0.7f,	 1.0f, 0.45f, 0.3f, 0.0f, 0.6f, 0.2f, // col 0
		0.7f, 0.7f,	 1.0f, 0.7f,  0.3f, 0.1f, 0.3f, 0.3f, // col 1
		1.0f, 1.0f,	 0.3f, 0.0f,  1.0f, 0.2f, 0.2f, 0.3f, // col 2
		0.5f, 0.15f, 0.3f, 0.4f,  0.2f, 0.4f, 0.4f, 0.4f, // col 3
		1.0f, 1.0f,	 0.9f, 0.2f,  0.4f, 1.0f, 0.3f, 0.5f, // col 4
		0.3f, 0.2f,	 0.0f, 1.0f,  1.0f, 1.0f, 0.3f, 0.6f, // col 5
		0.4f, 0.2f,	 0.0f, 1.0f,  0.2f, 0.4f, 0.4f, 0.4f, // col 6
		0.2f, 0.1f,	 0.3f, 0.4f,  0.3f, 0.2f, 0.8f, 0.2f, // col 7
		0.2f, 0.6f,	 0.0f, 0.9f,  0.3f, 0.1f, 0.3f, 0.3f, // col 8
		0.2f, 0.2f,	 0.0f, 0.3f,  0.3f, 0.8f, 0.2f, 0.2f, // col 9
	};
	const auto prand_ = std::vector<float>{
		// another random patch
		0.3f, 0.4f,	 0.4f, 0.8f, 0.2f, 0.1f, 0.3f, 0.3f, // col 0
		0.9f, 0.3f,	 1.0f, 0.7f, 0.3f, 0.9f, 0.9f, 0.2f, // col 1
		0.5f, 0.2f,	 0.9f, 1.0f, 1.0f, 0.5f, 0.2f, 0.1f, // col 2
		0.2f, 0.15f, 0.7f, 0.3f, 0.2f, 0.4f, 0.3f, 0.4f, // col 3
		0.3f, 0.4f,	 0.4f, 0.8f, 0.4f, 0.3f, 0.3f, 0.3f, // col 4
		0.6f, 1.0f,	 1.0f, 0.3f, 0.3f, 0.3f, 0.5f, 0.3f, // col 5
		0.6f, 0.3f,	 0.8f, 0.4f, 0.2f, 0.6f, 0.6f, 0.6f, // col 6
		0.8f, 0.2f,	 0.2f, 0.3f, 0.2f, 0.8f, 0.2f, 0.8f, // col 7
		0.3f, 0.8f,	 1.0f, 0.3f, 0.3f, 0.3f, 0.1f, 0.3f, // col 8
		0.3f, 0.1f,	 0.4f, 0.8f, 0.2f, 0.3f, 0.3f, 0.3f, // col 9
	};
	const auto ch6 = std::vector<float>(p0.size(), 0.0f);

	// Duplicates are intensional.
	auto data = p0; // 0
	data.insert(data.end(), p90.cbegin(), p90.cend()); // 1
	data.insert(data.end(), p180.cbegin(), p180.cend()); // 2
	data.insert(data.end(), prand.cbegin(), prand.cend()); // 3
	data.insert(data.end(), p180.cbegin(), p180.cend()); // 4
	data.insert(data.end(), prand.cbegin(), prand.cend()); // 5
	data.insert(data.end(), prand.cbegin(), prand.cend()); // 6
	data.insert(data.end(), prand_.cbegin(), prand_.cend()); // 7
	data.insert(data.end(), p270.cbegin(), p270.cend()); // 8
	data.insert(data.end(), ch6.cbegin(), ch6.cend()); // 9

	return device_patches<float>{data, 10_patches, 8_rows, 10_cols};
}

std::vector<size_t> clustering_solution()
{
	return std::vector<size_t>{1, 9, 7, 3};
}

// Distance matrix for lhs_patches x rhs_patches
void distance_matrix(const device_matrix<double>& dist_mat, double margin_low, double margin_high)
{
	REQUIRE(dist_mat.rows() == 4);
	REQUIRE(dist_mat.cols() == 4);

	CHECK(dist_mat.at(0_rows, 0_cols) == Approx{0.0}.margin(margin_low)); //  id - id
	CHECK(dist_mat.at(1_rows, 0_cols) == Approx{0.0}.margin(margin_low)); //  id - id
	CHECK(dist_mat.at(2_rows, 0_cols) > margin_high); //                      id - id_
	CHECK(dist_mat.at(3_rows, 0_cols) > margin_high); //                      id - id_

	CHECK(dist_mat.at(0_rows, 1_cols) == Approx{0.0}.margin(margin_low)); //  id  - rotation
	CHECK(dist_mat.at(1_rows, 1_cols) == Approx{0.0}.margin(margin_low)); //  id  - rotation
	CHECK(dist_mat.at(2_rows, 1_cols) > margin_high); //                      id_ - rotation
	CHECK(dist_mat.at(3_rows, 1_cols) > margin_high); //                      id_ - rotation

	CHECK(dist_mat.at(0_rows, 2_cols) > margin_high); //                     id  - scaling_
	CHECK(dist_mat.at(1_rows, 2_cols) > margin_high); //                     id  - scaling_
	CHECK(dist_mat.at(2_rows, 2_cols) == Approx{0.0}.margin(margin_low)); // id_ - scaling_
	CHECK(dist_mat.at(3_rows, 2_cols) == Approx{0.0}.margin(margin_low)); // id_ - scaling_

	CHECK(dist_mat.at(0_rows, 3_cols) > margin_high); // 					 id_ - zero
	CHECK(dist_mat.at(1_rows, 3_cols) > margin_high); // 					 id_ - zero
	CHECK(dist_mat.at(2_rows, 3_cols) > margin_high); // 					 id_ - zero
	CHECK(dist_mat.at(3_rows, 3_cols) > margin_high); // 					 id_ - zero
}
} // namespace data_similar