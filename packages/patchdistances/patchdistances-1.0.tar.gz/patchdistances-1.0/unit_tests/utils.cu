#include "hip/hip_runtime.h"
#include "utils.h"

#include "../include/device_matrix.h"
#include "../include/device_patches.h"
#include "../include/enums.h"
#include "../include/utils.h"

#include "../extern/gsl/gsl_util"

#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/inner_product.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/random.h>
#include <thrust/transform.h>

#include <hip/hip_complex.h>

#include <string>
#include <vector>

/*
**
*** Functions that return random matrices and patches using uniform distribution with values in
*** [0,1).
**
*/

template<typename Float>
struct rand_op
{
	rand_op(size_t off) : m{off} {}

	// Returns unique random numbers for unique n.
	__device__ Float operator()(size_t n)
	{
		auto rng = thrust::default_random_engine{};
		auto dist = thrust::uniform_real_distribution<Float>(0, 1);
		rng.discard(m + n);
		return dist(rng);
	}

  private:
	size_t m;
};

//! Returns a device_vector of random floats sampled from [0,1).
template<typename Float, typename = std::enable_if_t<std::is_floating_point<Float>::value>>
thrust::device_vector<Float> rand_vector(size_t size, Float /*tag*/)
{
	static auto offset = size_t{0}; // Ensures different output on each function call.
	auto rand_vec = thrust::device_vector<Float>(size);

	const auto begin = thrust::counting_iterator<size_t>{0};
	const auto end = thrust::counting_iterator<size_t>{size};
	thrust::transform(begin, end, rand_vec.begin(), rand_op<Float>{offset});

	offset += size;

	return rand_vec;
}

template<typename Float>
thrust::device_vector<vec3<Float>> rand_vector(size_t size, vec3<Float> /*tag*/)
{
	const auto ch0 = rand_vector(size, Float{});
	const auto ch1 = rand_vector(size, Float{});
	const auto ch2 = rand_vector(size, Float{});

	const auto ch0_ptr = ch0.data().get();
	const auto ch1_ptr = ch1.data().get();
	const auto ch2_ptr = ch2.data().get();

	auto data = thrust::device_vector<vec3<Float>>(ch0.size());
	const auto begin = thrust::counting_iterator<size_t>{0};
	const auto end = thrust::counting_iterator<size_t>{data.size()};
	thrust::transform(begin, end, data.begin(), [ch0_ptr, ch1_ptr, ch2_ptr] __device__(size_t i) {
		return vec3<Float>{ch0_ptr[i], ch1_ptr[i], ch2_ptr[i]};
	});

	return data;
}

device_matrix<float> rand_matrix(Size s, float tag)
{
	return device_matrix<float>{rand_vector(s.total(), tag), s};
}

device_matrix<double> rand_matrix(Size s, double tag)
{
	return device_matrix<double>{rand_vector(s.total(), tag), s};
}

device_matrix<vec3<float>> rand_matrix(Size s, vec3<float> tag)
{
	return device_matrix<vec3<float>>{rand_vector(s.total(), tag), s};
}

device_matrix<vec3<double>> rand_matrix(Size s, vec3<double> tag)
{
	return device_matrix<vec3<double>>{rand_vector(s.total(), tag), s};
}

device_patches<float> rand_patches(patch_index p, Size s, float tag)
{
	return device_patches<float>{rand_vector(p.value() * s.total(), tag), p, s};
}

device_patches<double> rand_patches(patch_index p, Size s, double tag)
{
	return device_patches<double>{rand_vector(p.value() * s.total(), tag), p, s};
}

device_patches<vec3<float>> rand_patches(patch_index p, Size s, vec3<float> tag)
{
	return device_patches<vec3<float>>{rand_vector(p.value() * s.total(), tag), p, s};
}

device_patches<vec3<double>> rand_patches(patch_index p, Size s, vec3<double> tag)
{
	return device_patches<vec3<double>>{rand_vector(p.value() * s.total(), tag), p, s};
}

/*
**
*** Catch2 checks, which compare matrices and patches. Eps is the expected L2 error (for each patch/
*** matrix) and margin its error margin.
**
*/

template<typename T>
struct L2 final
{
	__device__ double operator()(T lhs, T rhs) const noexcept
	{
		return static_cast<double>(sqrtf((lhs - rhs) * (lhs - rhs)));
	}
};

template<>
struct L2<vec3<float>> final
{
	__device__ double operator()(vec3<float> lhs, vec3<float> rhs) const noexcept
	{
		const auto x = lhs._1 - rhs._1;
		const auto y = lhs._2 - rhs._2;
		const auto z = lhs._3 - rhs._3;

		return norm3d(x, y, z);
	}
};

template<>
struct L2<vec3<double>> final
{
	__device__ double operator()(vec3<double> lhs, vec3<double> rhs) const noexcept
	{
		const auto x = lhs._1 - rhs._1;
		const auto y = lhs._2 - rhs._2;
		const auto z = lhs._3 - rhs._3;

		return norm3d(x, y, z);
	}
};

template<typename Complex>
constexpr __device__ double abs(const Complex& val) noexcept
{
	return sqrt(val.x * val.x + val.y * val.y + val.z * val.z);
}

template<>
struct L2<hipComplex> final
{
	__device__ double operator()(hipComplex lhs, hipComplex rhs) const noexcept
	{
		return hipCabsf(hipCsubf(lhs, rhs));
	}
};

template<>
struct L2<hipDoubleComplex> final
{
	__device__ double operator()(hipDoubleComplex lhs, hipDoubleComplex rhs) const noexcept
	{
		return hipCabs(hipCsub(lhs, rhs));
	}
};

template<>
struct L2<vec3<hipComplex>> final
{
	__device__ double operator()(vec3<hipComplex> lhs, vec3<hipComplex> rhs) const noexcept
	{
		const auto x = hipCabsf(hipCsubf(lhs._1, rhs._1));
		const auto y = hipCabsf(hipCsubf(lhs._2, rhs._2));
		const auto z = hipCabsf(hipCsubf(lhs._3, rhs._3));

		return norm3d(x, y, z);
	}
};

template<>
struct L2<vec3<hipDoubleComplex>> final
{
	__device__ double operator()(vec3<hipDoubleComplex> lhs,
								 vec3<hipDoubleComplex> rhs) const noexcept
	{
		const auto x = hipCabs(hipCsub(lhs._1, rhs._1));
		const auto y = hipCabs(hipCsub(lhs._2, rhs._2));
		const auto z = hipCabs(hipCsub(lhs._3, rhs._3));

		return norm3d(x, y, z);
	}
};

//! A template function that returns the L2 error of two matrices.
/*!
 * \param lhs Left hand side matrix.
 * \param rhs Right hand side matrix.
 * \return L2 error between lhs and rhs.
 */
template<typename T>
double l2_error(const device_matrix<T>& lhs, const device_matrix<T>& rhs)
{
	Expects(lhs.size() == rhs.size());
	Expects(lhs.ld() == rhs.ld());
	Expects(lhs.col_maj() == rhs.col_maj());

	return sqrt(thrust::inner_product(lhs.cbegin(), lhs.cend(), rhs.cbegin(), 0.0,
									  thrust::plus<double>(), L2<T>{}));
}

//! A template function that returns the L2 error of each pair of patches.
/*!
 * \param lhs Left hand side patches.
 * \param rhs Right hand side patches.
 * \return L2 error between lhs and rhs for each patch.
 */
template<typename T>
inline std::vector<double> l2_error(const device_patches<T>& lhs, const device_patches<T>& rhs)
{
	Expects(lhs.col_maj_patches() == rhs.col_maj_patches());
	Expects(lhs.size_patches() == rhs.size_patches());
	Expects(lhs.patch_count() == rhs.patch_count());

	auto errs = std::vector<double>(lhs.patch_count());
	for(size_t i = 0; i < errs.size(); i++)
	{
		const auto lhs_cbegin = lhs.cbegin(patch_index{i});
		const auto lhs_cend = lhs.cend(patch_index{i});
		const auto rhs_cbegin = rhs.cbegin(patch_index{i});
		errs.at(i) = sqrt(thrust::inner_product(lhs_cbegin, lhs_cend, rhs_cbegin, 0.0,
												thrust::plus<double>{}, L2<T>{}));
	}

	return errs;
}

template<typename T>
void is_equal_impl(const device_patches<T>& lhs, const device_patches<T>& rhs, double eps,
				   double margin)
{
	REQUIRE(lhs.patch_count() == lhs.patch_count());
	REQUIRE(lhs.size_patches() == rhs.size_patches());
	REQUIRE(lhs.col_maj_patches() == rhs.col_maj_patches());

	const auto errs = l2_error(lhs, rhs);
	for(std::size_t i = 0; i < errs.size(); i++)
	{
		const auto total_per_patch = gsl::narrow_cast<double>(lhs.total_per_patch());
		if(errs.at(i) / total_per_patch != Approx{eps}.margin(margin))
		{
			FAIL("The relative L2-error is too large, starting at patch no: "
				 << i << ". \nIs:     " << errs.at(i) / total_per_patch << "\neps:    " << eps
				 << "\nmargin: " << margin);
			break;
		}
	}
}

template<typename T>
void is_equal_impl(const device_matrix<T>& lhs, const device_matrix<T>& rhs, double eps,
				   double margin)
{
	REQUIRE(lhs.size() == rhs.size());
	REQUIRE(lhs.col_maj() == rhs.col_maj());

	const auto total = gsl::narrow_cast<double>(lhs.total());
	const auto err = l2_error(lhs, rhs) / total;
	if(err != Approx{eps}.margin(margin))
	{
		FAIL("The relative L2-error is too large.\nIs:     " << err << "\neps:    " << eps
															 << "\nmargin: " << margin);
	}
}

template<typename T>
void not_equal_impl(const device_matrix<T>& lhs, const device_matrix<T>& rhs, double eps,
					double margin)
{
	if(lhs.size() != rhs.size())
	{
		CHECK(lhs.size() != rhs.size());
		return;
	}
	if(lhs.col_maj() != rhs.col_maj())
	{
		CHECK(lhs.col_maj() != rhs.col_maj());
		return;
	}

	const auto total = gsl::narrow_cast<double>(lhs.total());
	CHECK(l2_error(lhs, rhs) / total != Approx{eps}.margin(margin));
}

void is_equal(const device_patches<float>& lhs, const device_patches<float>& rhs, double eps,
			  double margin)
{
	is_equal_impl(lhs, rhs, eps, margin);
}

void is_equal(const device_patches<double>& lhs, const device_patches<double>& rhs, double eps,
			  double margin)
{
	is_equal_impl(lhs, rhs, eps, margin);
}

void is_equal(const device_patches<vec3<float>>& lhs, const device_patches<vec3<float>>& rhs,
			  double eps, double margin)
{
	is_equal_impl(lhs, rhs, eps, margin);
}

void is_equal(const device_patches<vec3<double>>& lhs, const device_patches<vec3<double>>& rhs,
			  double eps, double margin)
{
	is_equal_impl(lhs, rhs, eps, margin);
}

void is_equal(const device_patches<hipComplex>& lhs, const device_patches<hipComplex>& rhs,
			  double eps, double margin)
{
	is_equal_impl(lhs, rhs, eps, margin);
}

void is_equal(const device_patches<hipDoubleComplex>& lhs,
			  const device_patches<hipDoubleComplex>& rhs, double eps, double margin)
{
	is_equal_impl(lhs, rhs, eps, margin);
}

void is_equal(const device_patches<vec3<hipComplex>>& lhs,
			  const device_patches<vec3<hipComplex>>& rhs, double eps, double margin)
{
	is_equal_impl(lhs, rhs, eps, margin);
}

void is_equal(const device_patches<vec3<hipDoubleComplex>>& lhs,
			  const device_patches<vec3<hipDoubleComplex>>& rhs, double eps, double margin)
{
	is_equal_impl(lhs, rhs, eps, margin);
}

void is_equal(const device_matrix<size_t>& lhs, const device_matrix<size_t>& rhs, double eps,
			  double margin)
{
	is_equal_impl(lhs, rhs, eps, margin);
}

void is_equal(const device_matrix<float>& lhs, const device_matrix<float>& rhs, double eps,
			  double margin)
{
	is_equal_impl(lhs, rhs, eps, margin);
}

void is_equal(const device_matrix<double>& lhs, const device_matrix<double>& rhs, double eps,
			  double margin)
{
	is_equal_impl(lhs, rhs, eps, margin);
}

void is_equal(const device_matrix<vec3<float>>& lhs, const device_matrix<vec3<float>>& rhs,
			  double eps, double margin)
{
	is_equal_impl(lhs, rhs, eps, margin);
}

void is_equal(const device_matrix<vec3<double>>& lhs, const device_matrix<vec3<double>>& rhs,
			  double eps, double margin)
{
	is_equal_impl(lhs, rhs, eps, margin);
}

void not_equal(const device_matrix<float>& lhs, const device_matrix<float>& rhs, double eps,
			   double margin)
{
	not_equal_impl(lhs, rhs, eps, margin);
}

void not_equal(const device_matrix<double>& lhs, const device_matrix<double>& rhs, double eps,
			   double margin)
{
	not_equal_impl(lhs, rhs, eps, margin);
}

void not_equal(const device_matrix<vec3<float>>& lhs, const device_matrix<vec3<float>>& rhs,
			   double eps, double margin)
{
	not_equal_impl(lhs, rhs, eps, margin);
}

void not_equal(const device_matrix<vec3<double>>& lhs, const device_matrix<vec3<double>>& rhs,
			   double eps, double margin)
{
	not_equal_impl(lhs, rhs, eps, margin);
}