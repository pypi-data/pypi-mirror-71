#include "hip/hip_runtime.h"
#include "../include/device_patches.h"
#include "../include/log_polar_transform.h"
#include "../include/sample.h"
#include "../include/vec3.h"

#include "../extern/gsl/gsl_util"

#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>

#include <cmath>

constexpr auto TwoPi = 6.283185307179586476925286766559005768394338798750211641949f;
constexpr auto Pi = 3.1415926535897932384626433832795028841971693993751058209749445923078164062f;

//! Returns radius for log polar interpolation for a given image size.
/*!
 * \param i_size
 * \param bool i_whole_image If true the radius covers the whole image, otherwise it is the largest
 * radius contained in the image.
 * \return Log-polar radius.
 */
float get_log_polar_radius(const Size& i_size, bool i_whole_image)
{
	if(i_whole_image)
	{
		// +1 to avoid too small radii, because of rounding errors.
		const auto a2 = gsl::narrow_cast<float>(i_size.cols() * i_size.cols() + 1) / 4.0f;
		const auto b2 = gsl::narrow_cast<float>(i_size.rows() * i_size.rows() + 1) / 4.0f;

		return sqrt(a2 + b2);
	}
	else
	{
		// Too small radii, because of rounding errors do not matter here.
		return gsl::narrow_cast<float>(std::min(i_size.cols(), i_size.rows())) * 0.5f;
	}
}

//! Log-polar interpolation of patches with a fixed radius of 2*pi.
/*!
 * \param i_patches Patches to interpolate.
 * \param i_M Rows of the interpolated patches.
 * \param i_N Columns of the interpolated patches.
 * \param i_embed If true the patches are embedded in a black background first.
 * Otherwise, the largest disk contained in the patches is interpolated.
 * \param i_transpose If true the output will be transposed.
 * \return Log-polar interpolated patches.
 */
template<typename T>
device_patches<T> log_polar_transform_impl(const device_patches<T>& i_patches, Rows i_N, Cols i_M,
										   bool i_embed, bool i_transpose)
{
	const auto R = get_log_polar_radius(i_patches.size_patches(), i_embed);
	const auto rows = gsl::narrow_cast<int>(i_patches.rows_patches());
	const auto cols = gsl::narrow_cast<int>(i_patches.cols_patches());
	const auto rows_2 = gsl::narrow_cast<float>(i_patches.rows_patches() - 1) / 2.0f;
	const auto cols_2 = gsl::narrow_cast<float>(i_patches.cols_patches() - 1) / 2.0f;
	const auto ld = i_patches.ld();
	const auto col_maj = i_patches.col_maj_patches();

	// Angular and radial sampling steps
	const auto N = i_N.value();
	const auto M = i_M.value();
	const auto d_rho = TwoPi / gsl::narrow_cast<float>(N);
	const auto d_theta = TwoPi / gsl::narrow_cast<float>(M);

	const auto rows_out = i_transpose ? M : N;
	const auto cols_out = i_transpose ? N : M;
	auto o_patches = device_patches<T>{patch_index{i_patches.patch_count()}, Rows{rows_out},
									   Cols{cols_out}, col_maj};
	const auto o_ld = o_patches.ld();
	auto o_ptr = o_patches.data().get();
	const auto i_ptr = i_patches.data().get();

	// Each thread computes one pixel of a log_polar patch.
	const auto out_first = thrust::counting_iterator<std::size_t>{0};
	const auto out_last = thrust::counting_iterator<std::size_t>{o_patches.total()};
	thrust::for_each(out_first, out_last, [=] __device__(std::size_t el) {
		const auto p_no = el / o_ld;
		const auto p_el = el - p_no * o_ld;

		const auto _col_maj = i_transpose ? !col_maj : col_maj;
		const auto n = _col_maj ? p_el % N : p_el / M;
		const auto m = _col_maj ? p_el / N : p_el % M;

		// To be able to use FFT in the similarity invariant distance, the radius must be exp(2*pi)!
		// Thus, after computing the beam, we have to scale it accordingly.
		const auto beam = exp(n * d_rho) / exp(TwoPi) * R;
		const auto angle = m * d_theta;
		const auto x = beam * cos(angle) + cols_2;
		const auto y = beam * sin(angle) + rows_2;

		o_ptr[el] = sample(i_ptr + p_no * ld, x, y, rows, cols, col_maj, interpolation_t::bicubic);
	});

	return o_patches;
}

//! Template function to check if any component of a vec3 is nan.
template<typename T>
__device__ constexpr bool isnan(const vec3<T>& val) noexcept
{
	return isnan(val._1) || isnan(val._2) || isnan(val._3);
}

//! Inverse log-polar interpolation of patches for a fixed log polar radius of 2*pi.
/*!
 * \param i_patches Patches to interpolate.
 * \param i_M Rows of the interpolated patches.
 * \param i_N Columns of the interpolated patches.
 * \param i_embed If true, the patches were embedded in a black background in the forward
 * transformation. Otherwise, the largest disk contained in the patches was interpolated.
 * \param i_transposed If true, the input is transposed.
 * \return Inverse log-polar interpolated patches.
 */
template<typename T>
device_patches<T> log_polar_inv_transform_impl(const device_patches<T>& i_patches, Rows i_Y,
											   Cols i_X, bool i_embed, bool i_transposed)
{
	const auto R = get_log_polar_radius(Size{i_Y, i_X}, i_embed);
	const auto rows = gsl::narrow_cast<int>(i_patches.rows_patches());
	const auto cols = gsl::narrow_cast<int>(i_patches.cols_patches());
	const auto ld = i_patches.ld();
	const auto ld_patches = i_patches.ld_patches();
	const auto col_maj = i_patches.col_maj_patches();

	const auto Y = i_Y.value();
	const auto X = i_X.value();
	const auto Y_2 = gsl::narrow_cast<float>(Y - 1) * 0.5f;
	const auto X_2 = gsl::narrow_cast<float>(X - 1) * 0.5f;

	// Angular and radial sampling steps
	const auto N = i_transposed ? i_patches.cols_patches() : i_patches.rows_patches();
	const auto M = i_transposed ? i_patches.rows_patches() : i_patches.cols_patches();
	const auto d_rho = TwoPi / gsl::narrow_cast<float>(N);
	const auto d_theta = TwoPi / gsl::narrow_cast<float>(M);

	auto o_patches = device_patches<T>{patch_index{i_patches.patch_count()}, i_Y, i_X, col_maj};
	const auto o_ld = o_patches.ld();
	auto o_ptr = o_patches.data().get();
	const auto i_ptr = i_patches.data().get();
	const auto o_p_total = o_patches.total();

	// Each thread computes one pixel of the inverse log_polar patch.
	const auto out_first = thrust::counting_iterator<std::size_t>{0};
	const auto out_last = thrust::counting_iterator<std::size_t>{o_p_total};
	thrust::for_each(out_first, out_last, [=] __device__(std::size_t el) {
		const auto p_no = el / o_ld;
		const auto p_el = (p_no + 1) * o_ld - el - 1;

		const auto x = static_cast<float>(col_maj ? p_el / Y : p_el % X);
		const auto y = static_cast<float>(col_maj ? p_el % Y : p_el / X);

		// Cave: the log polar transformation uses a fixed radius of exp(2*pi)!
		// Thus, n has to be scaled accordingly.
		const auto x_2 = x - X_2;
		const auto y_2 = y - Y_2;
		const auto m = (atan2(y_2, x_2) + Pi) / d_theta;
		const auto n = log(sqrt(x_2 * x_2 + y_2 * y_2) * exp(TwoPi) / R) / d_rho;
		const auto val =
			i_transposed
				? sample(i_ptr + p_no * ld, n, m, rows, cols, col_maj, interpolation_t::bicubic)
				: sample(i_ptr + p_no * ld, m, n, rows, cols, col_maj, interpolation_t::bicubic);

		o_ptr[el] = isnan(val) ? T{} : val; // center is nan (atan2)
	});

	return o_patches;
}

device_patches<float> log_polar_transform(const device_patches<float>& i_patches, Rows i_M,
										  Cols i_N, bool i_embed, bool i_transpose)
{
	return log_polar_transform_impl(i_patches, i_M, i_N, i_embed, i_transpose);
}

device_patches<double> log_polar_transform(const device_patches<double>& i_patches, Rows i_M,
										   Cols i_N, bool i_embed, bool i_transpose)
{
	return log_polar_transform_impl(i_patches, i_M, i_N, i_embed, i_transpose);
}

device_patches<vec3<float>> log_polar_transform(const device_patches<vec3<float>>& i_patches,
												Rows i_M, Cols i_N, bool i_embed, bool i_transpose)
{
	return log_polar_transform_impl(i_patches, i_M, i_N, i_embed, i_transpose);
}

device_patches<vec3<double>> log_polar_transform(const device_patches<vec3<double>>& i_patches,
												 Rows i_M, Cols i_N, bool i_embed, bool i_transpose)
{
	return log_polar_transform_impl(i_patches, i_M, i_N, i_embed, i_transpose);
}

device_patches<float> log_polar_inv_transform(const device_patches<float>& i_patches, Rows i_Y,
											  Cols i_X, bool i_embed, bool i_transposed)
{
	return log_polar_inv_transform_impl(i_patches, i_Y, i_X, i_embed, i_transposed);
}

device_patches<double> log_polar_inv_transform(const device_patches<double>& i_patches, Rows i_Y,
											   Cols i_X, bool i_embed, bool i_transposed)
{
	return log_polar_inv_transform_impl(i_patches, i_Y, i_X, i_embed, i_transposed);
}

device_patches<vec3<float>> log_polar_inv_transform(const device_patches<vec3<float>>& i_patches,
													Rows i_Y, Cols i_X, bool i_embed,
													bool i_transposed)
{
	return log_polar_inv_transform_impl(i_patches, i_Y, i_X, i_embed, i_transposed);
}

device_patches<vec3<double>> log_polar_inv_transform(const device_patches<vec3<double>>& i_patches,
													 Rows i_Y, Cols i_X, bool i_embed,
													 bool i_transposed)
{
	return log_polar_inv_transform_impl(i_patches, i_Y, i_X, i_embed, i_transposed);
}