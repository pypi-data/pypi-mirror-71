#include "hip/hip_runtime.h"
#include "../include/device_patches.h"
#include "../include/procrustes_solver.h"
#include "../include/unique_cublas_handle.h"
#include "../include/utils.h"
#include "../include/warp_perspective.h"

#include "../extern/gsl/gsl_assert"
#include "../extern/gsl/gsl_util"

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>

#include <utility>
#include <vector>

using uint = unsigned int;
using lli = long long int;
// using namespace impl;

/*
 * Thin wrappers for cublas functions.
 */

void gemm(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb, int m, int n,
		  int k, const float* alpha, const float* Aarray[], int lda, const float* Barray[], int ldb,
		  const float* beta, float* Carray[], int ldc, int batchCount)
{
	check_cublas(hipblasSgemmBatched(handle, transa, transb, m, n, k, alpha, Aarray, lda, Barray,
									ldb, beta, Carray, ldc, batchCount));
	check(hipDeviceSynchronize(), hipSuccess);
}

void gemm(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb, int m, int n,
		  int k, const double* alpha, const double* Aarray[], int lda, const double* Barray[],
		  int ldb, const double* beta, double* Carray[], int ldc, int batchCount)
{
	check_cublas(hipblasDgemmBatched(handle, transa, transb, m, n, k, alpha, Aarray, lda, Barray,
									ldb, beta, Carray, ldc, batchCount));
	check(hipDeviceSynchronize(), hipSuccess);
}

//! Matrix multiplication for patches.
/*!
 * \param i_alpha Scalar factor.
 * \param i_transA W/o transposing A.
 * \param i_A Patches.
 * \param i_transB W/o transposing B.
 * \param i_B Patches.
 * \param i_h Initialized cublas handle.
 * \return alpha * A[i] * B[i] + beta * C[i], where i denotes a patch.
 */
template<typename Float>
device_patches<Float> mat_mul(Float i_alpha, hipblasOperation_t i_transA,
							  const device_patches<Float>& i_A, hipblasOperation_t i_transB,
							  const device_patches<Float>& i_B, hipblasHandle_t i_h)
{
	Expects(i_A.col_maj_patches());
	Expects(i_B.col_maj_patches());
	Expects(i_A.patch_count() == i_B.patch_count());

	const auto rows_A = i_transA == HIPBLAS_OP_N ? i_A.rows_patches() : i_A.cols_patches();
	const auto cols_A = i_transA == HIPBLAS_OP_N ? i_A.cols_patches() : i_A.rows_patches();
	const auto cols_B = i_transB == HIPBLAS_OP_N ? i_B.cols_patches() : i_B.rows_patches();

	const auto beta = Float{0.0};
	const auto m = gsl::narrow<int>(rows_A);
	const auto n = gsl::narrow<int>(cols_B);
	const auto k = gsl::narrow<int>(cols_A);
	const auto lda = gsl::narrow<int>(i_A.ld_patches());
	const auto ldb = gsl::narrow<int>(i_B.ld_patches());
	const auto ldc = m;
	const auto batchCount = gsl::narrow<int>(i_A.patch_count());

	auto C = device_patches<Float>{patch_index{batchCount}, Rows{m}, Cols{n}};
	const auto A_stride = i_A.ld();
	const auto B_stride = i_B.ld();
	const auto C_stride = C.ld();
	const auto A_ptr = i_A.data().get();
	const auto B_ptr = i_B.data().get();
	const auto C_ptr = C.data().get();
	auto Aarray = thrust::device_vector<const Float*>(batchCount);
	auto Barray = thrust::device_vector<const Float*>(batchCount);
	auto Carray = thrust::device_vector<Float*>(batchCount);

	const auto begin = thrust::make_counting_iterator<size_t>(0);
	const auto end = thrust::make_counting_iterator<size_t>(batchCount);
	thrust::transform(begin, end, Aarray.begin(),
					  [A_ptr, A_stride] __device__(size_t pos) { return A_ptr + pos * A_stride; });
	thrust::transform(begin, end, Barray.begin(),
					  [B_ptr, B_stride] __device__(size_t pos) { return B_ptr + pos * B_stride; });
	thrust::transform(begin, end, Carray.begin(),
					  [C_ptr, C_stride] __device__(size_t pos) { return C_ptr + pos * C_stride; });

	gemm(i_h, i_transA, i_transB, m, n, k, &i_alpha, Aarray.data().get(), lda, Barray.data().get(),
		 ldb, &beta, Carray.data().get(), ldc, batchCount);

	return C;
}

//! A function that returns the first order moments for given image moments (not including zero-th
//! moments).
/*!
 * \param moments Image moments.
 * \return First order image moments (not including zero-th moments).
 */
template<typename Float>
device_patches<Float> get_first_order_moments(const device_patches<Float>& moments)
{
	Expects(moments.cols_patches() >= 3);
	Expects(moments.col_maj_patches());

	const auto p_count = moments.patch_count();
	const auto rows = moments.rows_patches();
	const auto m_stride = moments.ld();
	const auto m_ptr = moments.data().get();
	auto first_order_moments = device_patches<Float>{patch_index{p_count}, Rows{rows}, Cols{2}};
	auto fom_ptr = first_order_moments.data().get();

	const auto begin = thrust::make_counting_iterator<size_t>(0);
	const auto end = thrust::make_counting_iterator<size_t>(p_count);

	thrust::for_each(begin, end, [m_ptr, fom_ptr, m_stride, rows] __device__(size_t patch) {
		const auto m_off = patch * m_stride + rows;
		const auto fom_off = patch * 2 * rows;
		for(size_t i = 0; i < 2 * rows; i++)
		{
			fom_ptr[fom_off + i] = m_ptr[m_off + i];
		}
	});

	return first_order_moments;
}

//! Singular value decomposition for 2x2 matrices.
/*!
 * \param a Input matrix.
 * \param u Left hand side singular vector.
 * \param v Right hand side singular vector.
 */
template<typename Float>
__host__ __device__ void svd_2x2_impl(const Float* a, Float u[4], Float v[4])
{
	if(a[0] == 0.0 && a[1] == 0.0 && a[2] == 0.0 && a[3] == 0.0)
	{
		u[0] = 1.0;
		u[1] = 0.0;
		u[2] = 0.0;
		u[3] = 1.0;

		v[0] = 1.0;
		v[1] = 0.0;
		v[2] = 0.0;
		v[3] = 1.0;
	}
	else
	{
		const auto a0_a3 = (a[0] - a[3]) * (a[0] - a[3]);
		const auto a0pa3 = (a[0] + a[3]) * (a[0] + a[3]);
		const auto a1pa2 = (a[1] + a[2]) * (a[1] + a[2]);
		const auto a1_a2 = (a[1] - a[2]) * (a[1] - a[2]);

		const auto s0 = 0.5f * (sqrt(a0_a3 + a1pa2) + sqrt(a0pa3 + a1_a2));
		const auto s1 = abs(s0 - sqrt(a0_a3 + a1pa2));

		const auto y = 2 * (a[0] * a[1] + a[2] * a[3]);
		const auto x = a[0] * a[0] - a[1] * a[1] + a[2] * a[2] - a[3] * a[3];

		u[2] = (s0 > s1) ? -sin(0.5f * atan2(y, x)) : 0;
		u[0] = -sqrt(1 - u[2] * u[2]);
		u[1] = u[2];
		u[3] = -u[0];

		v[0] = (s0 != 0) ? (a[0] * u[0] + a[1] * u[2]) / s0 : 1;
		v[2] = (s0 != 0) ? (a[2] * u[0] + a[3] * u[2]) / s0 : 0;
		v[1] = (s1 != 0) ? (a[0] * u[1] + a[1] * u[3]) / s1 : -v[2];
		v[3] = (s1 != 0) ? (a[2] * u[1] + a[3] * u[3]) / s1 : v[0];
	}
}

__host__ __device__ void svd_2x2(const float* a, float u[4], float v[4])
{
	return svd_2x2_impl(a, u, v);
}

__host__ __device__ void svd_2x2(const double* a, double u[4], double v[4])
{
	return svd_2x2_impl(a, u, v);
}

//! Singular value decomposition for 2x2 matrices.
/*!
 * \param u Left hand side matrix.
 * \param v Right hand side matrix.
 * \param uv Matrix product u*v.
 */
template<typename Float>
__host__ __device__ void mat_mul_2x2_impl(const Float u[4], const Float v[4], Float* uv)
{
	// u, v, uv column major
	uv[0] = u[0] * v[0] + u[2] * v[1];
	uv[1] = u[1] * v[0] + u[3] * v[1];
	uv[2] = u[0] * v[2] + u[2] * v[3];
	uv[3] = u[1] * v[2] + u[3] * v[3];
}

__host__ __device__ void mat_mul_2x2(const float u[4], const float v[4], float* uv)
{
	return mat_mul_2x2_impl(u, v, uv);
}
__host__ __device__ void mat_mul_2x2(const double u[4], const double v[4], double* uv)
{
	return mat_mul_2x2_impl(u, v, uv);
}

//! CUDA kernel that computes the perspective transformations from a 2x2 patches.
/*!
 * \param A Patches of size 2x2.
 * \param transformations Output perspective transformations.
 * \param patch_size Size of the corresponding image patches.
 */
template<typename Float>
__global__ void procrustes_transforms_kernel(const Float* __restrict__ A,
											 float* __restrict__ transformations, Size patch_size)
{
	const auto off_in = blockIdx.x * 2 * 2;
	const auto off_out = blockIdx.x * 3 * 3;
	Float u[4];
	Float v[4];
	Float uv[4];

	svd_2x2(A + off_in, u, v);
	mat_mul_2x2(u, v, uv);

	// a = uv^t
	// uv has column major layout, a has row-major layout
	const auto a11 = static_cast<float>(uv[0]);
	const auto a12 = static_cast<float>(uv[2]);
	const auto a21 = static_cast<float>(uv[1]);
	const auto a22 = static_cast<float>(uv[3]);
	const auto b1 = 0.0f;
	const auto b2 = 0.0f;

	// Non invertible matrices have to be explicitly excluded.
	const auto det = a11 * a22 - a12 * a21;
	if(abs(det) > 0.001)
	{
		perspective_matrix(a11, a12, b1, a21, a22, b2, patch_size, transformations + off_out);
	}
	else // identity
	{
		perspective_matrix(1.0f, 0.0f, 0.0f, 0.0f, 1.0f, 0.0f, patch_size,
						   transformations + off_out);
	}
}

//! A function that computes the perspective transformations from a 2x2 patches.
/*!
 * \param A Patches of size 2x2.
 * \param patch_size Size of the corresponding image patches.
 * \return Perspective transformations.
 */
template<typename Float>
device_patches<float> procrustes_transforms(const device_patches<Float>& A, const Size& patch_size)
{
	Expects(A.col_maj_patches());
	Expects(A.rows_patches() == 2);
	Expects(A.cols_patches() == 2);

	const auto blockDim = dim3{1};
	const auto gridDim = dim3{gsl::narrow<uint>(A.patch_count())};
	const auto row_maj = false;
	auto transformations =
		device_patches<float>{patch_index{A.patch_count()}, 3_rows, 3_cols, row_maj};

	procrustes_transforms_kernel<<<gridDim, blockDim>>>(A.data().get(),
														transformations.data().get(), patch_size);
	check(hipDeviceSynchronize(), hipSuccess);

	return transformations;
}

//! A function computing the affine transformations of image patches, based on the image moments.
/*!
 * \param lhs Image moments.
 * \param rhs Image moments.
 * \param patch_size Size of the images corresponding to the moments.
 * \param hipblas.h Initialized cublas handle.
 * \return Perspective transformation for each pair of patches.
 */
template<typename Float>
device_patches<float> procrustes_solver_impl(const device_patches<Float>& lhs,
											 const device_patches<Float>& rhs,
											 const Size& patch_size, hipblasHandle_t hipblas.h)
{
	const auto lhs1 = get_first_order_moments(lhs);
	const auto rhs1 = get_first_order_moments(rhs);

	const auto alpha = Float{1.0};
	auto lhs1t_rhs1 = mat_mul(alpha, HIPBLAS_OP_T, rhs1, HIPBLAS_OP_N, lhs1, hipblas.h); // rhs1^t*lhs1

	return procrustes_transforms(lhs1t_rhs1, patch_size);
}

device_patches<float> procrustes_solver(const device_patches<float>& lhs,
										const device_patches<float>& rhs, const Size& patch_size,
										hipblasHandle_t hipblas.h)
{
	return procrustes_solver_impl(lhs, rhs, patch_size, hipblas.h);
}

device_patches<float> procrustes_solver(const device_patches<double>& lhs,
										const device_patches<double>& rhs, const Size& patch_size,
										hipblasHandle_t hipblas.h)
{
	return procrustes_solver_impl(lhs, rhs, patch_size, hipblas.h);
}