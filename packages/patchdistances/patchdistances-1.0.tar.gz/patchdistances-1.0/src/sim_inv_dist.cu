#include "hip/hip_runtime.h"
#include "../include/sim_inv_dist.h"
#include "../src/sim_inv_dist_impl.h"

#include <array>
#include <utility>

device_matrix<double> sim_inv_dist::distance_matrix(const device_patches<float>& i_patches0,
													const device_patches<float>& i_patches1,
													const sid_params& i_params)
{
	return pimpl->distance_matrix(i_patches0, i_patches1, i_params);
}

device_matrix<double> sim_inv_dist::distance_matrix(const device_patches<double>& i_patches0,
													const device_patches<double>& i_patches1,
													const sid_params& i_params)
{
	return pimpl->distance_matrix(i_patches0, i_patches1, i_params);
}

device_matrix<double> sim_inv_dist::distance_matrix(const device_patches<vec3<float>>& i_patches0,
													const device_patches<vec3<float>>& i_patches1,
													const sid_params& i_params)
{
	return pimpl->distance_matrix(i_patches0, i_patches1, i_params);
}

device_matrix<double> sim_inv_dist::distance_matrix(const device_patches<vec3<double>>& i_patches0,
													const device_patches<vec3<double>>& i_patches1,
													const sid_params& i_params)
{
	return pimpl->distance_matrix(i_patches0, i_patches1, i_params);
}

std::pair<device_patches<float>, std::vector<size_t>>
	sim_inv_dist::greedy_k_center(const device_patches<float>& i_patches, size_t i_clusters,
								  size_t i_first, const sid_params& i_params)
{
	return pimpl->greedy_k_center(i_patches, i_clusters, i_first, i_params);
}

std::pair<device_patches<double>, std::vector<size_t>>
	sim_inv_dist::greedy_k_center(const device_patches<double>& i_patches, size_t i_clusters,
								  size_t i_first, const sid_params& i_params)
{
	return pimpl->greedy_k_center(i_patches, i_clusters, i_first, i_params);
}

std::pair<device_patches<vec3<float>>, std::vector<size_t>>
	sim_inv_dist::greedy_k_center(const device_patches<vec3<float>>& i_patches, size_t i_clusters,
								  size_t i_first, const sid_params& i_params)
{
	return pimpl->greedy_k_center(i_patches, i_clusters, i_first, i_params);
}

std::pair<device_patches<vec3<double>>, std::vector<size_t>>
	sim_inv_dist::greedy_k_center(const device_patches<vec3<double>>& i_patches, size_t i_clusters,
								  size_t i_first, const sid_params& i_params)
{
	return pimpl->greedy_k_center(i_patches, i_clusters, i_first, i_params);
}

std::pair<device_matrix<float>, device_matrix<size_t>>
	sim_inv_dist::reconstruct(const device_patches<float>& i_data_patches,
							  const device_patches<float>& i_labels, const Size& i_img_size,
							  rec_t i_type, const sid_params& i_params,
							  interpolation_t i_interpolation)
{
	return pimpl->reconstruct(i_data_patches, i_labels, i_img_size, i_type, i_params,
							  i_interpolation);
}

std::pair<device_matrix<double>, device_matrix<size_t>>
	sim_inv_dist::reconstruct(const device_patches<double>& i_data_patches,
							  const device_patches<double>& i_labels, const Size& i_img_size,
							  rec_t i_type, const sid_params& i_params,
							  interpolation_t i_interpolation)
{
	return pimpl->reconstruct(i_data_patches, i_labels, i_img_size, i_type, i_params,
							  i_interpolation);
}

std::pair<device_matrix<vec3<float>>, device_matrix<size_t>>
	sim_inv_dist::reconstruct(const device_patches<vec3<float>>& i_data_patches,
							  const device_patches<vec3<float>>& i_labels, const Size& i_img_size,
							  rec_t i_type, const sid_params& i_params,
							  interpolation_t i_interpolation)
{
	return pimpl->reconstruct(i_data_patches, i_labels, i_img_size, i_type, i_params,
							  i_interpolation);
}

std::pair<device_matrix<vec3<double>>, device_matrix<size_t>>
	sim_inv_dist::reconstruct(const device_patches<vec3<double>>& i_data_patches,
							  const device_patches<vec3<double>>& i_labels, const Size& i_img_size,
							  rec_t i_type, const sid_params& i_params,
							  interpolation_t i_interpolation)
{
	return pimpl->reconstruct(i_data_patches, i_labels, i_img_size, i_type, i_params,
							  i_interpolation);
}

std::pair<device_matrix<float>, device_matrix<size_t>> sim_inv_dist::reconstruct_w_translation(
	const device_patches<float>& i_data_patches, const device_patches<float>& i_labels,
	const Size& i_img_size, rec_t i_type, const sid_params& i_params,
	interpolation_t i_interpolation)
{
	return pimpl->reconstruct_w_translation(i_data_patches, i_labels, i_img_size, i_type, i_params,
											i_interpolation);
}

std::pair<device_matrix<double>, device_matrix<size_t>> sim_inv_dist::reconstruct_w_translation(
	const device_patches<double>& i_data_patches, const device_patches<double>& i_labels,
	const Size& i_img_size, rec_t i_type, const sid_params& i_params,
	interpolation_t i_interpolation)
{
	return pimpl->reconstruct_w_translation(i_data_patches, i_labels, i_img_size, i_type, i_params,
											i_interpolation);
}

std::pair<device_matrix<vec3<float>>, device_matrix<size_t>>
	sim_inv_dist::reconstruct_w_translation(const device_patches<vec3<float>>& i_data_patches,
											const device_patches<vec3<float>>& i_labels,
											const Size& i_img_size, rec_t i_type,
											const sid_params& i_params,
											interpolation_t i_interpolation)
{
	return pimpl->reconstruct_w_translation(i_data_patches, i_labels, i_img_size, i_type, i_params,
											i_interpolation);
}

std::pair<device_matrix<vec3<double>>, device_matrix<size_t>>
	sim_inv_dist::reconstruct_w_translation(const device_patches<vec3<double>>& i_data_patches,
											const device_patches<vec3<double>>& i_labels,
											const Size& i_img_size, rec_t i_type,
											const sid_params& i_params,
											interpolation_t i_interpolation)
{
	return pimpl->reconstruct_w_translation(i_data_patches, i_labels, i_img_size, i_type, i_params,
											i_interpolation);
}

device_patches<float>
	sim_inv_dist::perspective_transformations(const device_patches<float>& i_data_patches,
											  const device_patches<float>& i_labels,
											  const sid_params& i_params)
{
	return pimpl->perspective_transformations(i_data_patches, i_labels, i_params);
}

device_patches<float>
	sim_inv_dist::perspective_transformations(const device_patches<double>& i_data_patches,
											  const device_patches<double>& i_labels,
											  const sid_params& i_params)
{
	return pimpl->perspective_transformations(i_data_patches, i_labels, i_params);
}

device_patches<float>
	sim_inv_dist::perspective_transformations(const device_patches<vec3<float>>& i_data_patches,
											  const device_patches<vec3<float>>& i_labels,
											  const sid_params& i_params)
{
	return pimpl->perspective_transformations(i_data_patches, i_labels, i_params);
}

device_patches<float>
	sim_inv_dist::perspective_transformations(const device_patches<vec3<double>>& i_data_patches,
											  const device_patches<vec3<double>>& i_labels,
											  const sid_params& i_params)
{
	return pimpl->perspective_transformations(i_data_patches, i_labels, i_params);
}

device_patches<hipComplex> sim_inv_dist::afmt(const device_patches<float>& i_patches, float i_sigma)
{
	return pimpl->afmt(i_patches, i_sigma);
}

device_patches<hipDoubleComplex> sim_inv_dist::afmt(const device_patches<double>& i_patches,
												   float i_sigma)
{
	return pimpl->afmt(i_patches, i_sigma);
}

device_patches<vec3<hipComplex>> sim_inv_dist::afmt(const device_patches<vec3<float>>& i_patches,
												   float i_sigma)
{
	return pimpl->afmt(i_patches, i_sigma);
}

device_patches<vec3<hipDoubleComplex>>
	sim_inv_dist::afmt(const device_patches<vec3<double>>& i_patches, float i_sigma)
{
	return pimpl->afmt(i_patches, i_sigma);
}

/*
**
*** sim_inv_dist::impl
**
*/

//! Initializes cuff plan and returns fft output size.
/*!
 * cuFTT data layout
 * input[b * idist + (x * inembed[1] + y) * istride]
 * output[b * odist + (x * onembed[1] + y) * ostride]
 * \param i_M Input dimension of fft. (Square input expected.)
 * \param i_patch_count Batch number of the fft.
 * \param i_single If true single precision, otherwise double precision.
 * \o_stride Stride of the out memory elements. (For the input 1 is expected.)
 */
Size sim_inv_dist::impl::init(size_t i_M, size_t i_patch_count, bool i_single, size_t o_stride)
{
	Expects(i_M > 0);
	Expects(i_patch_count > 0);

	// Cufft stores n/2 + 1 elements in the columns.
	const auto int_M = gsl::narrow<int>(i_M);
	const auto int_N = gsl::narrow<int>(i_M);
	const auto ostride = gsl::narrow<int>(o_stride);
	const auto int_pc = gsl::narrow<int>(i_patch_count);

	if(i_M != m_M || i_patch_count != m_patch_count || i_single != m_single)
	{
		// TODO: Is it necessary to get a new plan?
		m_plan.reset();

		// fft dimension
		// cave: contiguous dimension last
		auto N = std::array<int, 2>{int_M, int_N};
		const auto batch = int_pc;

		// storage dimension input
		auto inembed = std::array<int, 2>{int_N, int_M};
		const auto istride = 1;
		const auto idist = int_M * int_N;

		// storage dimension output
		// Cufft computes only the unique coefficients.
		auto onembed = std::array<int, 2>{int_M, int_N / 2 + 1};
		const auto odist = ostride * (int_N / 2 + 1) * (int_M);

		const auto type = i_single ? HIPFFT_R2C : HIPFFT_D2Z;

		constexpr auto Rank = 2;
		auto worksize = size_t{}; // unused
		check(hipfftMakePlanMany(m_plan.get(), Rank, N.data(), inembed.data(), istride, idist,
								onembed.data(), ostride, odist, type, batch, &worksize),
			  HIPFFT_SUCCESS);

		m_M = i_M;
		m_patch_count = i_patch_count;
		m_single = i_single;
	}

	// Cufft computes only the unique coefficients.
	return Size{Rows{int_N / 2 + 1}, Cols{int_M}};
}

// Synchronization is done with scaleafmt().
inline void cufftExec(hipfftHandle plan, hipfftReal* idata, hipComplex* odata)
{
	check_cufft(hipfftExecR2C(plan, idata, odata));
}

// Synchronization is done with scaleafmt().
inline void cufftExec(hipfftHandle plan, hipfftDoubleReal* idata, hipDoubleComplex* odata)
{
	check_cufft(hipfftExecD2Z(plan, idata, odata));
}

// The struct is needed since __device__ lambdas are not supported in private members.
template<typename Float>
struct afmt_func_struct_1D final
{
  public:
	//! Constructor.
	/*!
	 * \param i_ptr Log-polar transformed patches (transposed! and continuous).
	 * \param o_ptr Output patches of the same size, read for the fft (continuous).
	 * \param i_rows Rows per patch.
	 * \param i_cols Columns per patches.
	 * \param i_sigma Sigma parameter from afmt.
	 */
	afmt_func_struct_1D(const Float* i_ptr, Float* o_ptr, size_t i_rows, size_t i_cols,
						float i_sigma) noexcept
		: in_ptr{i_ptr}, out_ptr{o_ptr}, sigma{i_sigma}, patches_size{i_rows * i_cols},
		  rows{i_rows}, rowsT{static_cast<float>(i_rows)}, colsT{static_cast<float>(i_cols)}
	{
	}

	//! Use the fft symmetry to put the origin in the frequency space in the center of the fft
	//! output  and multiply afmt related constants.
	/*!
	 * \param el Index of the input element from 0 to size of patches * number of patches.
	 */
	__device__ void operator()(std::size_t el) const noexcept
	{
		// Cave: The log polar image is expected to be transposed!
		const auto el_patch = el % patches_size;
		const auto m = static_cast<float>(el_patch % rows);
		const auto n = static_cast<float>(el_patch / rows);

		out_ptr[el] =
			TwoPi / rowsT * exp(TwoPi * sigma * n / colsT) * pow(-1.0f, n + m) * in_ptr[el];
	}

  private:
	const Float* in_ptr;
	Float* out_ptr;
	float sigma;
	size_t patches_size;
	size_t rows;
	float rowsT;
	float colsT;
};

// The struct is needed since __device__ lambdas are not supported in private members.
template<typename Float>
struct afmt_func_struct_3D final
{
  public:
	//! Constructor.
	/*!
	 * \param i_ptr Log-polar transformed patches (transposed! and continuous).
	 * \param o_ptr0 Output patches of the first channel of the same size, read for the fft
	 * (continuous).
	 * \param o_ptr1 Output patches of the second channel of the same size, read for the fft
	 * (continuous).
	 * \param o_ptr2 Output patches of the third channel of the same size, read for the fft
	 * (continuous).
	 * \param i_rows Rows per patch.
	 * \param i_cols Columns per patches.
	 * \param i_sigma Sigma parameter from afmt.
	 */
	afmt_func_struct_3D(const vec3<Float>* i_ptr, Float* o_ptr0, Float* o_ptr1, Float* o_ptr2,
						size_t i_rows, size_t i_cols, float i_sigma) noexcept
		: in_ptr{i_ptr}, out_ptr0{o_ptr0}, out_ptr1{o_ptr1}, out_ptr2{o_ptr2}, sigma{i_sigma},
		  patches_size{i_rows * i_cols}, rows{i_rows}, rowsT{static_cast<float>(i_rows)},
		  colsT{static_cast<float>(i_cols)}
	{
	}

	//! Use the fft symmetry to put the origin in the frequency space in the center of the fft
	//! output  and multiply afmt related constants.
	/*!
	 * \param el Index of the input element from 0 to size of patches * number of patches.
	 */
	__device__ void operator()(std::size_t el) const noexcept
	{
		// Cave: The log polar image is expected to be transposed!
		const auto el_patch = el % patches_size;
		const auto m = static_cast<float>(el_patch % rows);
		const auto n = static_cast<float>(el_patch / rows);

		const auto res =
			TwoPi / rowsT * exp(TwoPi * sigma * n / colsT) * pow(-1.0f, n + m) * in_ptr[el];
		out_ptr0[el] = res._1;
		out_ptr1[el] = res._2;
		out_ptr2[el] = res._3;
	}

  private:
	const vec3<Float>* in_ptr;
	Float* out_ptr0;
	Float* out_ptr1;
	Float* out_ptr2;
	float sigma;
	size_t patches_size;
	size_t rows;
	float rowsT;
	float colsT;
};

//! A function that pre-processes image patches to reduce the analytic Fourier-Mellin
//! transformation to a fast Fourier transform (FFT).
/*!
 * \param i_patches Image patches.
 * \param i_sigma Fourier-Mellin-Coefficient.
 * \return Pre-processed images.
 */
template<typename Float>
device_patches<Float> pre_fft(const device_patches<Float>& i_patches, float i_sigma)
{
	Expects(i_patches.col_maj_patches());
	Expects(i_patches.rows_patches() % 2 == 0); // Necessary for shift to center frequency.
	Expects(i_patches.rows_patches() == i_patches.cols_patches());
	// Rows == Cols: Necessary for fft, since the patches need to be transposed.

	// Shift the zero frequency to the center and multiply afmt related constants.
	const auto in_ptr = i_patches.data().get();
	const auto K = i_patches.rows_patches();
	const auto V = i_patches.cols_patches();
	const auto patches_size = i_patches.total_per_patch();
	const auto patches_rows = i_patches.rows_patches();
	const auto patches_first = thrust::counting_iterator<std::size_t>{0};
	const auto patches_last = thrust::counting_iterator<std::size_t>{i_patches.total()};

	const auto pc = patch_index{i_patches.patch_count()};
	const auto size = i_patches.size_patches();
	auto out = device_patches<Float>{pc, size};
	auto out_ptr = out.data().get();

	thrust::for_each(patches_first, patches_last,
					 afmt_func_struct_1D<Float>{in_ptr, out_ptr, K, V, i_sigma});

	return out;
}

/*!
 * \overlaod
 */
template<typename Float>
std::array<device_patches<Float>, 3> pre_fft(const device_patches<vec3<Float>>& i_patches,
											 float i_sigma)
{
	Expects(i_patches.col_maj_patches());
	Expects(i_patches.rows_patches() % 2 == 0); // Necessary for shift to center frequency.
	Expects(i_patches.rows_patches() == i_patches.cols_patches());
	// Rows == Cols: Necessary for fft, since the patches need to be transposed.

	// Shift the zero frequency to the center and multiply afmt related constants.
	const auto in_ptr = i_patches.data().get();
	const auto K = i_patches.rows_patches();
	const auto V = i_patches.cols_patches();
	const auto patches_size = i_patches.total_per_patch();
	const auto patches_rows = i_patches.rows_patches();
	const auto patches_first = thrust::counting_iterator<std::size_t>{0};
	const auto patches_last = thrust::counting_iterator<std::size_t>{i_patches.total()};

	using T = device_patches<Float>;
	const auto pc = patch_index{i_patches.patch_count()};
	const auto size = i_patches.size_patches();
	auto out = std::array<T, 3>{T{pc, size}, T{pc, size}, T{pc, size}};
	auto out0 = out[0].data().get();
	auto out1 = out[1].data().get();
	auto out2 = out[2].data().get();

	thrust::for_each(patches_first, patches_last,
					 afmt_func_struct_3D<Float>{in_ptr, out0, out1, out2, K, V, i_sigma});

	return out;
}

device_patches<hipComplex> sim_inv_dist::impl::afmt(const device_patches<float>& i_patches,
												   float i_sigma)
{
	auto pre_patches = pre_fft(i_patches, i_sigma);

	const auto single_prec = true;
	const auto pc = pre_patches.patch_count();
	const auto o_stride = 1;
	auto afmts_size = init(pre_patches.rows_patches(), pc, single_prec, o_stride);
	auto afmts = device_patches<hipComplex>{patch_index{pc}, afmts_size};

	cufftExec(m_plan.get(), pre_patches.data().get(), afmts.data().get());
	check(hipDeviceSynchronize(), hipSuccess);

	return afmts;
}

device_patches<hipDoubleComplex> sim_inv_dist::impl::afmt(const device_patches<double>& i_patches,
														 float i_sigma)
{
	auto pre_patches = pre_fft(i_patches, i_sigma);

	const auto double_prec = false;
	const auto pc = pre_patches.patch_count();
	const auto o_stride = 1;
	auto afmts_size = init(pre_patches.rows_patches(), pc, double_prec, o_stride);
	auto afmts = device_patches<hipDoubleComplex>{patch_index{pc}, afmts_size};

	cufftExec(m_plan.get(), pre_patches.data().get(), afmts.data().get());
	check(hipDeviceSynchronize(), hipSuccess);

	return afmts;
}

device_patches<vec3<hipComplex>>
	sim_inv_dist::impl::afmt(const device_patches<vec3<float>>& i_patches, float i_sigma)
{
	// Due to alignment(?), the input data has to be copied first; the output need not be copied.
	auto pre_patches = pre_fft(i_patches, i_sigma);
	const auto single_prec = true;
	const auto pc = i_patches.patch_count();
	const auto o_stride = 3;
	const auto afmts_size = init(i_patches.rows_patches(), pc, single_prec, o_stride);

	auto out = device_patches<vec3<hipComplex>>{patch_index{pc}, afmts_size};
	auto out_ptr = reinterpret_cast<hipComplex*>(out.data().get());

	cufftExec(m_plan.get(), pre_patches[0].data().get(), out_ptr);
	cufftExec(m_plan.get(), pre_patches[1].data().get(), out_ptr + 1);
	cufftExec(m_plan.get(), pre_patches[2].data().get(), out_ptr + 2);
	check(hipDeviceSynchronize(), hipSuccess);

	return out;
}

device_patches<vec3<hipDoubleComplex>>
	sim_inv_dist::impl::afmt(const device_patches<vec3<double>>& i_patches, float i_sigma)
{
	// Due to alignment(?), the input data has to be copied first; the output need not be copied.
	auto pre_patches = pre_fft(i_patches, i_sigma);
	const auto double_prec = false;
	const auto pc = i_patches.patch_count();
	const auto o_stride = 3;
	const auto afmts_size = init(i_patches.rows_patches(), pc, double_prec, o_stride);

	auto out = device_patches<vec3<hipDoubleComplex>>{patch_index{pc}, afmts_size};
	auto out_ptr = reinterpret_cast<hipDoubleComplex*>(out.data().get());

	cufftExec(m_plan.get(), pre_patches[0].data().get(), out_ptr);
	cufftExec(m_plan.get(), pre_patches[1].data().get(), out_ptr + 1);
	cufftExec(m_plan.get(), pre_patches[2].data().get(), out_ptr + 2);
	check(hipDeviceSynchronize(), hipSuccess);

	return out;
}