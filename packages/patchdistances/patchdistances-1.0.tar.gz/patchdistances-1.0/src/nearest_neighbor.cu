#include "hip/hip_runtime.h"
#include "../include/device_matrix.h"
#include "../include/nearest_neighbor.h"

#include "../extern/gsl/gsl_assert"

std::pair<device_matrix<size_t>, device_matrix<double>>
	nearest_neighbor_w_distances(const device_matrix<double>& i_dist_mat, const Size& i_image_size,
								 bool i_min)
{
	Expects(i_dist_mat.col_maj());
	Expects(i_dist_mat.rows() == i_image_size.total());

	const auto mat_ptr = i_dist_mat.data().get();
	const auto ld = i_dist_mat.ld();
	const auto cols = i_dist_mat.cols();

	const auto begin = thrust::counting_iterator<size_t>{0};
	const auto end = begin + i_dist_mat.rows();

	auto indices = device_matrix<size_t>{i_image_size};
	auto distances = device_matrix<double>{i_image_size};
	auto ind_ptr = indices.data().get();
	auto dist_ptr = distances.data().get();

	if(i_min) // nearest neighbor = minimum of a row
	{
		thrust::for_each(begin, end, [=] __device__(size_t row) {
			auto min = mat_ptr[row];
			auto ind = size_t{0};

			for(size_t i = 1; i < cols; i++)
			{
				const auto val = mat_ptr[row + i * ld];
				if(val < min)
				{
					min = val;
					ind = i;
				}
			}

			ind_ptr[row] = ind;
			dist_ptr[row] = min;
		});
	}
	else // nearest neighbor = maximum of a row
	{
		thrust::for_each(begin, end, [=] __device__(size_t row) {
			auto max = mat_ptr[row];
			auto ind = size_t{0};

			for(size_t i = 1; i < cols; i++)
			{
				const auto val = mat_ptr[row + i * ld];
				if(val > max)
				{
					max = val;
					ind = i;
				}
			}

			ind_ptr[row] = ind;
			dist_ptr[row] = max;
		});
	}

	return std::make_pair(std::move(indices), std::move(distances));
}

device_matrix<size_t> nearest_neighbor(const device_matrix<double>& i_dist_mat,
									   const Size& i_image_size, bool i_min)
{
	return std::get<0>(nearest_neighbor_w_distances(i_dist_mat, i_image_size, i_min));
}