#include "hip/hip_runtime.h"
#include "../include/determinant.h"
#include "../include/device_patches.h"
#include "../include/utils.h"

#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>

using uint = unsigned int;

//! Templated CUDA kernel to compute the determinant of an affine transformation from patch moments.
/*!
 * blockDim: 1, gridDim: Tf patches, Tg patches
 * The input and output memory is not allowed to overlap.
 * \param i_Tf A set of image moments matrices T.
 * \param i_Tg A set of image moments ymatrices T.
 * \param o_det Determinants of the affine transformation associated to the pair of moments i_Tf[i],
 * i_Tg[i].
 * \parma i_rows Rows of the matrices T.
 * \param i_stride Stride between the patches of Tf, Tg.
 */
template<typename Float>
__global__ void det_kernel(const Float* __restrict i_Tf, const Float* __restrict i_Tg,
						   Float* __restrict o_det, uint i_rows, uint i_stride)
{
	const auto off = blockIdx.x * i_stride;

	auto sum_fg = Float{};
	auto sum_gg = Float{};
	for(uint i = 0; i < i_rows; i++)
	{
		const auto f = i_Tf[off + i];
		const auto g = i_Tg[off + i];
		sum_fg += f * g;
		sum_gg += g * g;
	}

	o_det[blockIdx.x] = sum_fg / sum_gg;
}

//! Templated function to compute the determinant of an affine transformation from patch moments.
/*!
 * \param i_Tf A set of image moment matrices T.
 * \param i_Tg A set of image moment matrices T.
 * \return Determinants of the affine transformation associated to the pair of moments i_Tf[i],
 * i_Tg[i].
 */
template<typename Float>
thrust::device_vector<Float> determinant_impl(const device_patches<Float>& i_Tf,
											  const device_patches<Float>& i_Tg)
{
	Expects(i_Tf.patch_count() == i_Tg.patch_count());
	Expects(i_Tf.ld() == i_Tg.ld());

	auto dets = thrust::device_vector<Float>{i_Tf.patch_count()};
	const auto blockDim = dim3{1};
	const auto gridDim = dim3{gsl::narrow<uint>(i_Tf.patch_count())};
	const auto rows = gsl::narrow_cast<uint>(i_Tf.rows_patches());
	const auto stride = gsl::narrow_cast<uint>(i_Tf.ld());

	det_kernel<<<gridDim, blockDim>>>(i_Tf.data().get(), i_Tg.data().get(), dets.data().get(), rows,
									  stride);

	check(hipDeviceSynchronize(), hipSuccess);

	return dets;
}

thrust::device_vector<float> determinant(const device_patches<float>& i_Tf,
										 const device_patches<float>& i_Tg)
{
	return determinant_impl(i_Tf, i_Tg);
}

thrust::device_vector<double> determinant(const device_patches<double>& i_Tf,
										  const device_patches<double>& i_Tg)
{
	return determinant_impl(i_Tf, i_Tg);
}