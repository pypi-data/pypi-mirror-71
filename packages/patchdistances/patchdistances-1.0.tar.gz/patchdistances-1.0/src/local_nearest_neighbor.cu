#include "hip/hip_runtime.h"
#include "../include/device_matrix.h"
#include "../include/device_patches.h"
#include "../include/local_nearest_neighbor.h"
#include "../include/nearest_neighbor.h"

#include "../extern/gsl/gsl_assert"
#include "../extern/gsl/gsl_util"

#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>

#include <cstddef>

labeling local_nearest_neighbor(const device_matrix<double>& i_dist_mat,
								const Size& i_patches_shape, const Size& i_patch_size)
{
	Expects(i_patch_size.rows() % 2 == 1);
	Expects(i_patch_size.cols() % 2 == 1);

	const auto labels_distances = nearest_neighbor_w_distances(i_dist_mat, i_patches_shape);

	// Signed values are used to avoid unsigned underflow in the loops (See min(0, ...)).
	const auto begin = thrust::counting_iterator<int>{0};
	const auto end = begin + std::get<0>(labels_distances).total();

	// Safe because of small dimensions.
	const auto rows = gsl::narrow_cast<int>(std::get<0>(labels_distances).rows());
	const auto cols = gsl::narrow_cast<int>(std::get<0>(labels_distances).cols());
	const auto size = std::get<0>(labels_distances).size();

	const auto row_radius = gsl::narrow_cast<int>(i_patch_size.rows()) / 2;
	const auto col_radius = gsl::narrow_cast<int>(i_patch_size.cols()) / 2;

	auto labels = labeling{device_matrix<size_t>{size}, device_matrix<size_t>{size},
						   device_matrix<labeling::point>{size}};
	auto labels_ptr = labels.labels.data().get();
	auto patches_ptr = labels.patches.data().get();
	auto offsets_ptr = labels.offsets.data().get();
	const auto nn_labels_ptr = std::get<0>(labels_distances).data().get();
	const auto nn_dist_ptr = std::get<1>(labels_distances).data().get();

	// Iterate over every element in the nearest neighbor labeling and find the smallest distance in
	// the neighborhood.
	thrust::for_each(begin, end, [=] __device__(int ind) {
		const auto row = ind % rows;
		const auto col = ind / rows;

		// Given a pixel (center of an image patch). We consider all translations of a label
		// (patches), such that the label still coincides with the pixel. Additionally, near the
		// image border translation is limited, since those pixel do not correlate to image patches
		// (=are not the center of an image patch). Otherwise, after translation these may not be
		// covered anymore by a label.

		// Possible translation
		const auto row_t = min(min(row, rows - 1 - row), row_radius);
		const auto col_t = min(min(col, cols - 1 - col), col_radius);

		const auto row_begin = max(0, row - row_t);
		const auto row_end = min(rows, row + 1 + row_t);
		const auto col_begin = max(0, col - col_t);
		const auto col_end = min(cols, col + 1 + col_t);

		// Default is the coordinate of the current patch (no translation).
		auto off_rows = 0;
		auto off_cols = 0;
		auto off_ind = row + col * rows;
		auto min = nn_dist_ptr[off_ind];

		for(int j = col_begin; j < col_end; j++)
		{
			for(int i = row_begin; i < row_end; i++)
			{
				const auto loop_ind = i + j * rows;
				const auto val = nn_dist_ptr[loop_ind];
				if(val < min)
				{
					off_rows = i - row;
					off_cols = j - col;
					off_ind = loop_ind;
					min = val;
				}
			}
		}

		labels_ptr[ind] = nn_labels_ptr[off_ind];
		patches_ptr[ind] = off_ind;
		offsets_ptr[ind] = thrust::make_pair(off_cols, off_rows); // (x,y)
	});

	return labels;
}