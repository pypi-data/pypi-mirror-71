#include "hip/hip_runtime.h"
#include "../include/device_matrix.h"
#include "../include/imed.h"
#include "../include/utils.h"

#include "../extern/gsl/gsl_util"

#include <hip/hip_runtime.h>
#include <>

using uint = unsigned int;
constexpr auto SMEM_SIZE = 16 * 1024;

//! Templated CUDA kernel to compute the Convolution Standardized Transform (CFT) of patches.
/*!
 * gridDim: patch count; blockDim: patch size; smem: (patch_cols + 4) * (patch_rows + 4)
 * Each block computes a patch and each thread a pixel of a patch.
 * \param i_patches Input patches.
 * \param o_patches Convolved patches of the same size with boundary condition zero.
 * \param i_rows Rows of each patch i_patches.
 * \param i_cols
 * Columns of each patch i_patches.
 */
template<typename T>
__global__ void imed_cft_kernel_shared(const T* __restrict__ i_patches, T* __restrict__ o_patches,
									   uint i_rows, uint i_cols)
{
	// The convolution kernel is the Kronecker product of [0.0053, 0.2171, 0.5519, 0.2171, 0.0053].
	// Entries of the kernel.
	constexpr auto k00 = 2.809e-5;
	constexpr auto k10 = 1.15063e-3;
	constexpr auto k20 = 2.92507e-3;
	constexpr auto k30 = k10;
	constexpr auto k40 = k00;

	constexpr auto k01 = k10;
	constexpr auto k11 = 4.713241e-02;
	constexpr auto k21 = 1.1981749e-01;
	constexpr auto k31 = k11;
	constexpr auto k41 = k01;

	constexpr auto k02 = k20;
	constexpr auto k12 = k21;
	constexpr auto k22 = 3.0459361e-01;
	constexpr auto k32 = k12;
	constexpr auto k42 = k02;

	constexpr auto k03 = k30;
	constexpr auto k13 = k31;
	constexpr auto k23 = k32;
	constexpr auto k33 = k13;
	constexpr auto k43 = k03;

	constexpr auto k04 = k00;
	constexpr auto k14 = k10;
	constexpr auto k24 = k20;
	constexpr auto k34 = k30;
	constexpr auto k44 = k40;

	extern __shared__ __align__(alignof(T)) unsigned char smem[];
	T* s = reinterpret_cast<T*>(smem);

	// Load the patch including zero padding to shared memory.
	const auto padded_rows = i_rows + 4;
	const auto padded_cols = i_cols + 4;
	const auto padded_size = padded_rows * padded_cols;
	const auto idx = threadIdx.x + blockDim.x * blockIdx.x;
	const auto off_patch = blockIdx.x * i_rows * i_cols;

	for(uint i = threadIdx.x; i < padded_size; i += blockDim.x)
	{
		const auto padded_row = i % (i_rows + 4);
		const auto padded_col = i / (i_rows + 4);

		if(padded_row >= 2 && padded_row < i_rows + 2 && padded_col >= 2 && padded_col < i_cols + 2)
		{
			s[i] = i_patches[off_patch + padded_row - 2 + (padded_col - 2) * i_rows];
		}
		else
		{
			s[i] = T{0.0};
		}
	}
	__syncthreads();

	// Center coordinates of the current thread.
	// Upper left corner of the padded size.
	const auto row = threadIdx.x % i_rows;
	const auto col = threadIdx.x / i_rows;

	// Offset for each column.
	const auto col0 = row + col * padded_rows;
	const auto col1 = col0 + padded_rows;
	const auto col2 = col1 + padded_rows;
	const auto col3 = col2 + padded_rows;
	const auto col4 = col3 + padded_rows;

	const auto val00 = s[col0];
	const auto val10 = s[col0 + 1];
	const auto val20 = s[col0 + 2];
	const auto val30 = s[col0 + 3];
	const auto val40 = s[col0 + 4];

	const auto val01 = s[col1];
	const auto val11 = s[col1 + 1];
	const auto val21 = s[col1 + 2];
	const auto val31 = s[col1 + 3];
	const auto val41 = s[col1 + 4];

	const auto val02 = s[col2];
	const auto val12 = s[col2 + 1];
	const auto val22 = s[col2 + 2];
	const auto val32 = s[col2 + 3];
	const auto val42 = s[col2 + 4];

	const auto val03 = s[col3];
	const auto val13 = s[col3 + 1];
	const auto val23 = s[col3 + 2];
	const auto val33 = s[col3 + 3];
	const auto val43 = s[col3 + 4];

	const auto val04 = s[col4];
	const auto val14 = s[col4 + 1];
	const auto val24 = s[col4 + 2];
	const auto val34 = s[col4 + 3];
	const auto val44 = s[col4 + 4];

	// Column sums of the convolution, taking into account the symmetry of the kernel.
	const auto col0sum = val00 * k00 + val10 * k10 + val20 * k20 + val30 * k30 + val40 * k40;
	const auto col1sum = val01 * k01 + val11 * k11 + val21 * k21 + val31 * k31 + val41 * k41;
	const auto col2sum = val02 * k02 + val12 * k12 + val22 * k22 + val32 * k32 + val42 * k42;
	const auto col3sum = val03 * k03 + val13 * k13 + val23 * k23 + val33 * k33 + val43 * k43;
	const auto col4sum = val04 * k04 + val14 * k14 + val24 * k24 + val34 * k34 + val44 * k44;

	// It is safe to write, since loads are fenced by __syncthreads();
	o_patches[idx] = col0sum + col1sum + col2sum + col3sum + col4sum;
}

//! Templated CUDA kernel to compute the Convolution Standardized Transform (CFT) of patches.
/*!
 * gridDim: patch count, patch rows, patch columns
 * Each block computes a patch and each thread a pixel of a patch.
 * This kernel is slow and only used for large images.
 * \param i_patches Input patches.
 * \param o_patches Convolved patches of the same size with boundary condition zero.
 * Columns of each patch i_patches.
 */
template<typename T>
__global__ void imed_cft_kernel(const T* __restrict__ i_patches, T* __restrict__ o_patches)
{
	// The convolution kernel is the Kronecker product of [0.0053, 0.2171, 0.5519, 0.2171, 0.0053].
	// Entries of the kernel.
	constexpr auto k00 = 2.809e-5;
	constexpr auto k10 = 1.15063e-3;
	constexpr auto k20 = 2.92507e-3;
	constexpr auto k30 = k10;
	constexpr auto k40 = k00;

	constexpr auto k01 = k10;
	constexpr auto k11 = 4.713241e-02;
	constexpr auto k21 = 1.1981749e-01;
	constexpr auto k31 = k11;
	constexpr auto k41 = k01;

	constexpr auto k02 = k20;
	constexpr auto k12 = k21;
	constexpr auto k22 = 3.0459361e-01;
	constexpr auto k32 = k12;
	constexpr auto k42 = k02;

	constexpr auto k03 = k30;
	constexpr auto k13 = k31;
	constexpr auto k23 = k32;
	constexpr auto k33 = k13;
	constexpr auto k43 = k03;

	constexpr auto k04 = k00;
	constexpr auto k14 = k10;
	constexpr auto k24 = k20;
	constexpr auto k34 = k30;
	constexpr auto k44 = k40;

	const auto patch_no = blockIdx.x;
	const auto row_u = blockIdx.y;
	const auto col_u = blockIdx.z;
	const auto rows_u = gridDim.y;
	const auto cols_u = gridDim.z;
	const auto patch_size = rows_u * cols_u;
	const auto patch_idx = row_u + col_u * rows_u;
	const auto global_idx = patch_idx + patch_no * patch_size;
	const auto s = i_patches + patch_no * patch_size;

	// Offset for each column.
	const auto row = static_cast<int>(row_u); // Casts are safe, numbers are small
	const auto col = static_cast<int>(col_u);
	const auto rows = static_cast<int>(rows_u);
	const auto cols = static_cast<int>(cols_u);
	// Upper left corner of the kernel size.
	const auto col0 = row - 2 + (col - 2) * rows;
	const auto col1 = col0 + rows;
	const auto col2 = col1 + rows;
	const auto col3 = col2 + rows;
	const auto col4 = col3 + rows;

	const auto val00 = row >= 2 && col >= 2 ? s[col0] : T{0.0};
	const auto val10 = row >= 1 && col >= 2 ? s[col0 + 1] : T{0.0};
	const auto val20 = col >= 2 ? s[col0 + 2] : T{0.0};
	const auto val30 = row < rows - 1 && col >= 2 ? s[col0 + 3] : T{0.0};
	const auto val40 = row < rows - 2 && col >= 2 ? s[col0 + 4] : T{0.0};

	const auto val01 = row >= 2 && col >= 1 ? s[col1] : T{0.0};
	const auto val11 = row >= 1 && col >= 1 ? s[col1 + 1] : T{0.0};
	const auto val21 = col >= 1 ? s[col1 + 2] : T{0.0};
	const auto val31 = row < rows - 1 && col >= 1 ? s[col1 + 3] : T{0.0};
	const auto val41 = row < rows - 2 && col >= 1 ? s[col1 + 4] : T{0.0};

	const auto val02 = row >= 2 ? s[col2] : T{0.0};
	const auto val12 = row >= 1 ? s[col2 + 1] : T{0.0};
	const auto val22 = s[col2 + 2];
	const auto val32 = row < rows - 1 ? s[col2 + 3] : T{0.0};
	const auto val42 = row < rows - 2 ? s[col2 + 4] : T{0.0};

	const auto val03 = row >= 2 && col < cols - 1 ? s[col3] : T{0.0};
	const auto val13 = row >= 1 && col < cols - 1 ? s[col3 + 1] : T{0.0};
	const auto val23 = col < cols - 1 ? s[col3 + 2] : T{0.0};
	const auto val33 = row < rows - 1 && col < cols - 1 ? s[col3 + 3] : T{0.0};
	const auto val43 = row < rows - 2 && col < cols - 1 ? s[col3 + 4] : T{0.0};

	const auto val04 = row >= 2 && col < cols - 2 ? s[col4] : T{0.0};
	const auto val14 = row >= 1 && col < cols - 2 ? s[col4 + 1] : T{0.0};
	const auto val24 = col < cols - 2 ? s[col4 + 2] : T{0.0};
	const auto val34 = row < rows - 1 && col < cols - 2 ? s[col4 + 3] : T{0.0};
	const auto val44 = row < rows - 2 && col < cols - 2 ? s[col4 + 4] : T{0.0};

	// Column sums of the convolution, taking into account the symmetry of the kernel.
	const auto col0sum = val00 * k00 + val10 * k10 + val20 * k20 + val30 * k30 + val40 * k40;
	const auto col1sum = val01 * k01 + val11 * k11 + val21 * k21 + val31 * k31 + val41 * k41;
	const auto col2sum = val02 * k02 + val12 * k12 + val22 * k22 + val32 * k32 + val42 * k42;
	const auto col3sum = val03 * k03 + val13 * k13 + val23 * k23 + val33 * k33 + val43 * k43;
	const auto col4sum = val04 * k04 + val14 * k14 + val24 * k24 + val34 * k34 + val44 * k44;

	o_patches[global_idx] = col0sum + col1sum + col2sum + col3sum + col4sum;
}

template<typename T>
device_patches<T> imed_cft_impl(const device_patches<T>& i_patches)
{
	Expects(i_patches.col_maj_patches());

	const auto in_ptr = i_patches.data().get();
	const auto rows = gsl::narrow<uint>(i_patches.rows_patches());
	const auto cols = gsl::narrow<uint>(i_patches.cols_patches());
	const auto count = gsl::narrow<uint>(i_patches.patch_count());
	const auto smem = (rows + 4) * (cols + 4) * sizeof(T);
	auto out = device_patches<T>{patch_index{count}, i_patches.size_patches()};
	auto out_ptr = out.data().get();

	if(smem <= SMEM_SIZE)
	{
		const auto gridDim = dim3{count};
		const auto blockDim = dim3{rows * cols};
		imed_cft_kernel_shared<<<gridDim, blockDim, smem>>>(in_ptr, out_ptr, rows, cols);
	}
	else
	{
		const auto gridDim = dim3{count, rows, cols};
		const auto blockDim = dim3{1};
		imed_cft_kernel<<<gridDim, blockDim>>>(in_ptr, out_ptr);
	}

	check(hipDeviceSynchronize(), hipSuccess);

	return out;
}

device_patches<float> imed_cft(const device_patches<float>& i_patches)
{
	return imed_cft_impl(i_patches);
}

device_patches<double> imed_cft(const device_patches<double>& i_patches)
{
	return imed_cft_impl(i_patches);
}

device_patches<vec3<float>> imed_cft(const device_patches<vec3<float>>& i_patches)
{
	return imed_cft_impl(i_patches);
}

device_patches<vec3<double>> imed_cft(const device_patches<vec3<double>>& i_patches)
{
	return imed_cft_impl(i_patches);
}