#include "hip/hip_runtime.h"
#include "../include/device_patches.h"
#include "../include/enums.h"
#include "../include/sample.h"
#include "../include/utils.h"
#include "../include/vec3.h"
#include "../include/warp_perspective.h"

#include <hip/hip_runtime.h>

using uint = unsigned int;

/*
**
*** Implementation for patches
**
*/

template<typename FloatVec>
__device__ void warp_perspective_px_device(const FloatVec* __restrict__ input,
										   const float* __restrict__ A,
										   FloatVec* __restrict__ output, int px, int rows,
										   int cols, bool col_maj, interpolation_t i_type)
{
	const auto x = static_cast<float>(col_maj ? px / rows : px % cols);
	const auto y = static_cast<float>(col_maj ? px % rows : px / cols);

	// Compute homogeneous coordinate (x,y,1) for the sampled pixel.
	const auto z_in = A[6] * x + A[7] * y + A[8];
	const auto x_in = (A[0] * x + A[1] * y + A[2]) / z_in;
	const auto y_in = (A[3] * x + A[4] * y + A[5]) / z_in;

	output[px] = sample(input, x_in, y_in, rows, cols, col_maj, i_type);
}

template<typename FloatVec>
__global__ void warp_perspective_patches_kernel(const FloatVec* __restrict__ input,
												const float* __restrict__ transforms,
												FloatVec* __restrict__ output, int rows, int cols,
												size_t patch_count, bool col_maj,
												interpolation_t i_type)
{
	const auto px = static_cast<int>(threadIdx.x); // safe because of max blockDim.x
	const auto in = input + blockIdx.x * rows * cols;
	const auto A = transforms + blockIdx.x * 3 * 3;
	auto out = output + blockIdx.x * rows * cols;

	extern __shared__ __align__(alignof(FloatVec)) unsigned char smem[];
	FloatVec* s = reinterpret_cast<FloatVec*>(smem);

	// Patch size is equal to block size.
	s[threadIdx.x] = in[threadIdx.x];
	__syncthreads();

	warp_perspective_px_device(s, A, out, px, rows, cols, col_maj, i_type);
}

template<typename FloatVec>
device_patches<FloatVec> warp_perspective_impl(const device_patches<FloatVec>& i_patches,
											   const device_patches<float>& i_transforms,
											   interpolation_t i_type)
{
	Expects(i_patches.patch_count() == i_transforms.patch_count());
	Expects(i_transforms.rows_patches() == 3);
	Expects(i_transforms.cols_patches() == 3);
	Expects(i_transforms.col_maj_patches() == false);

	constexpr auto MAX_SMEM_SIZE = 16 * 1024;
	const auto p_total = gsl::narrow<uint>(i_patches.total_per_patch());
	const auto smem = p_total * sizeof(FloatVec);
	if(smem > MAX_SMEM_SIZE)
	{
		throw std::runtime_error{"Only patches with a size of up to " +
								 std::to_string(MAX_SMEM_SIZE) + "bytes are supported."};
	}

	constexpr auto MAX_BLOCK_SIZE = 16 * 1024;
	if(p_total > MAX_BLOCK_SIZE)
	{
		throw std::runtime_error{"Only patches with a size of up to " +
								 std::to_string(MAX_BLOCK_SIZE) + "px are supported."};
	}

	const auto rows = gsl::narrow<int>(i_patches.rows_patches());
	const auto cols = gsl::narrow<int>(i_patches.cols_patches());
	const auto p_count = gsl::narrow<uint>(i_patches.patch_count());
	const auto col_maj = i_patches.col_maj_patches();
	auto out = device_patches<FloatVec>{patch_index{p_count}, i_patches.size_patches(), col_maj};

	// Each thread computes one pixel of a patch.
	const auto blockDim = dim3{p_total};
	const auto gridDim = dim3{p_count};

	warp_perspective_patches_kernel<<<gridDim, blockDim, smem>>>(
		i_patches.data().get(), i_transforms.data().get(), out.data().get(), rows, cols, p_count,
		col_maj, i_type);
	check(hipDeviceSynchronize(), hipSuccess);

	return out;
}

/*
**
*** Implementation for matrix
**
*/

template<typename FloatVec>
__device__ void warp_perspective_device(const FloatVec* __restrict__ input,
										const float* __restrict__ A, FloatVec* __restrict__ output,
										int rows, int cols, bool col_maj, interpolation_t i_type)
{
	if(col_maj)
	{
		for(int x = 0; x < cols; x++)
		{
			for(int y = 0; y < rows; y++)
			{
				// Compute homogeneous coordinate (x,y,1) for the sampled pixel.
				const auto z_in = A[6] * x + A[7] * y + A[8];
				const auto x_in = (A[0] * x + A[1] * y + A[2]) / z_in;
				const auto y_in = (A[3] * x + A[4] * y + A[5]) / z_in;

				output[y + x * rows] = sample(input, x_in, y_in, rows, cols, col_maj, i_type);
			}
		}
	}
	else
	{
		for(int y = 0; y < rows; y++)
		{
			for(int x = 0; x < cols; x++)
			{
				// Compute homogeneous coordinate (x,y,1) for the sampled pixel.
				const auto z_in = A[6] * x + A[7] * y + A[8];
				const auto x_in = (A[0] * x + A[1] * y + A[2]) / z_in;
				const auto y_in = (A[3] * x + A[4] * y + A[5]) / z_in;

				output[x + y * cols] = sample(input, x_in, y_in, rows, cols, col_maj, i_type);
			}
		}
	}
}

template<typename FloatVec>
__global__ void warp_perspective_matrix_kernel(const FloatVec* __restrict__ input,
											   const float* __restrict__ transforms,
											   FloatVec* __restrict__ output, int rows, int cols,
											   size_t patch_count, bool col_maj,
											   interpolation_t i_type)
{
	const auto patch = threadIdx.x + blockIdx.x * blockDim.x;
	const auto smem_size = rows * cols;

	extern __shared__ __align__(alignof(FloatVec)) unsigned char smem[];
	FloatVec* s = reinterpret_cast<FloatVec*>(smem);

	for(auto i = threadIdx.x; i < smem_size; i += blockDim.x)
	{
		s[i] = input[i];
	}
	__syncthreads();

	if(patch < patch_count)
	{
		const auto A = transforms + patch * 3 * 3;
		auto out = output + patch * rows * cols;

		warp_perspective_device(s, A, out, rows, cols, col_maj, i_type);
	}
}

template<typename FloatVec>
device_patches<FloatVec> warp_perspective_impl(const device_matrix<FloatVec>& i_matrix,
											   const device_patches<float>& i_transforms,
											   interpolation_t i_type)
{
	Expects(i_transforms.rows_patches() == 3);
	Expects(i_transforms.cols_patches() == 3);
	Expects(i_transforms.col_maj_patches() == false);

	constexpr auto MAX_SMEM_SIZE = 16 * 1024;
	const auto smem = i_matrix.total() * sizeof(FloatVec);
	if(smem > MAX_SMEM_SIZE)
	{
		throw std::runtime_error{"Only matrices with a size of up to " +
								 std::to_string(MAX_SMEM_SIZE) + "bytes are supported."};
	}

	const auto rows = gsl::narrow<int>(i_matrix.rows());
	const auto cols = gsl::narrow<int>(i_matrix.cols());
	const auto p_count = gsl::narrow<uint>(i_transforms.patch_count());
	const auto col_maj = i_matrix.col_maj();
	auto out = device_patches<FloatVec>{patch_index{p_count}, i_matrix.size(), col_maj};

	// The threads are divided into large blocks to facilitate shared memory and reduce bank
	// conflicts when loading the input.
	const auto blockDim = dim3{256}; // Max blockDim.
	const auto gridDim = dim3{p_count / 256 + 1};

	warp_perspective_matrix_kernel<<<gridDim, blockDim, smem>>>(
		i_matrix.data().get(), i_transforms.data().get(), out.data().get(), rows, cols, p_count,
		col_maj, i_type);

	check(hipDeviceSynchronize(), hipSuccess);

	return out;
}

/*
**
*** Specializations
**
*/

device_patches<float> warp_perspective(const device_patches<float>& i_patches,
									   const device_patches<float>& i_transforms,
									   interpolation_t i_type)
{
	return warp_perspective_impl(i_patches, i_transforms, i_type);
}

device_patches<double> warp_perspective(const device_patches<double>& i_patches,
										const device_patches<float>& i_transforms,
										interpolation_t i_type)
{
	return warp_perspective_impl(i_patches, i_transforms, i_type);
}

device_patches<vec3<float>> warp_perspective(const device_patches<vec3<float>>& i_patches,
											 const device_patches<float>& i_transforms,
											 interpolation_t i_type)
{
	return warp_perspective_impl(i_patches, i_transforms, i_type);
}

device_patches<vec3<double>> warp_perspective(const device_patches<vec3<double>>& i_patches,
											  const device_patches<float>& i_transforms,
											  interpolation_t i_type)
{
	return warp_perspective_impl(i_patches, i_transforms, i_type);
}

device_patches<float> warp_perspective(const device_matrix<float>& i_matrix,
									   const device_patches<float>& i_transforms,
									   interpolation_t i_type)
{
	return warp_perspective_impl(i_matrix, i_transforms, i_type);
}

device_patches<double> warp_perspective(const device_matrix<double>& i_matrix,
										const device_patches<float>& i_transforms,
										interpolation_t i_type)
{
	return warp_perspective_impl(i_matrix, i_transforms, i_type);
}

device_patches<vec3<float>> warp_perspective(const device_matrix<vec3<float>>& i_patches,
											 const device_patches<float>& i_transforms,
											 interpolation_t i_type)
{
	return warp_perspective_impl(i_patches, i_transforms, i_type);
}

device_patches<vec3<double>> warp_perspective(const device_matrix<vec3<double>>& i_patches,
											  const device_patches<float>& i_transforms,
											  interpolation_t i_type)
{
	return warp_perspective_impl(i_patches, i_transforms, i_type);
}