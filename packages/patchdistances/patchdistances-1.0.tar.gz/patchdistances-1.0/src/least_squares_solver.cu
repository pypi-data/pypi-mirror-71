#include "hip/hip_runtime.h"
#include "../include/determinant.h"
#include "../include/device_patches.h"
#include "../include/least_squares_solver.h"
#include "../include/unique_cublas_handle.h"
#include "../include/utils.h"
#include "../include/warp_perspective.h"

#include "../extern/gsl/gsl_assert"
#include "../extern/gsl/gsl_util"

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/transform.h>

using uint = unsigned int;

inline __device__ float my_quiet_nan()
{
	return __int_as_float(0x7fffffff);
}

/*
 * Thin wrappers for cublas functions.
 */

void gels(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, int nrhs, float* Aarray[],
		  int lda, float* Carray[], int ldc, int batchSize)
{
	auto info = int{1};
	check_cublas(hipblasSgelsBatched(handle, trans, m, n, nrhs, Aarray, lda, Carray, ldc, &info,
									nullptr, batchSize));
	check(hipDeviceSynchronize(), hipSuccess);

	Ensures(info == 0);
}

void gels(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, int nrhs, double* Aarray[],
		  int lda, double* Carray[], int ldc, int batchSize)
{
	auto info = int{1};
	check_cublas(hipblasDgelsBatched(handle, trans, m, n, nrhs, Aarray, lda, Carray, ldc, &info,
									nullptr, batchSize));
	check(hipDeviceSynchronize(), hipSuccess);

	Ensures(info == 0);
}

//! Least squares for overdetermined systems, A * X = B.
/*!
 * \param lhs Matrix A.
 * \param rhs Matrix B.
 * \param handle Initialized cublas handle.
 * \return Matrix X.
 */
template<typename Float>
device_patches<Float> least_squares(const device_patches<Float>& lhs,
									const device_patches<Float>& rhs, hipblasHandle_t handle)
{
	Expects(lhs.col_maj_patches());
	Expects(rhs.col_maj_patches());
	Expects(lhs.rows_patches() == rhs.rows_patches());
	Expects(lhs.rows_patches() >= lhs.cols_patches());
	Expects(rhs.rows_patches() >= rhs.cols_patches());

	// The casts are safe, since the dimensions are small enough.
	const auto m = gsl::narrow_cast<int>(lhs.rows_patches());
	const auto n = gsl::narrow_cast<int>(lhs.cols_patches());
	const auto nrhs = gsl::narrow_cast<int>(rhs.cols_patches());
	const auto lda = gsl::narrow_cast<int>(lhs.ld_patches());
	const auto ldc = gsl::narrow_cast<int>(rhs.ld_patches());
	const auto batchSize = gsl::narrow_cast<int>(lhs.patch_count());

	// Solutions will overwrite inputs.
	auto _lhs = lhs;
	auto _rhs = rhs;
	const auto _lhs_ptr = _lhs.data().get();
	const auto _rhs_ptr = _rhs.data().get();
	const auto lhs_stride = lhs.ld();
	const auto rhs_stride = rhs.ld();

	// Gels needs device array of device pointer to each patch.
	auto _lhs_ptr_vec = thrust::device_vector<Float*>(batchSize);
	auto _rhs_ptr_vec = thrust::device_vector<Float*>(batchSize);

	const auto begin = thrust::make_counting_iterator<size_t>(0);
	const auto end = thrust::make_counting_iterator<size_t>(batchSize);

	thrust::transform(
		begin, end, _lhs_ptr_vec.begin(),
		[_lhs_ptr, lhs_stride] __device__(size_t pos) { return _lhs_ptr + pos * lhs_stride; });
	thrust::transform(
		begin, end, _rhs_ptr_vec.begin(),
		[_rhs_ptr, rhs_stride] __device__(size_t pos) { return _rhs_ptr + pos * rhs_stride; });

	gels(handle, HIPBLAS_OP_N, m, n, nrhs, _lhs_ptr_vec.data().get(), lda, _rhs_ptr_vec.data().get(),
		 ldc, batchSize);

	return _rhs;
}

//! Templated CUDA kernel that extracts the affine transformation from the least squares solution
//! of least_squares.
/*!
 * blockDim: 1, gridDim: patch count
 * \param i_Dt Solved linear systems from least squares.
 * The solution starts in the second column
 * D^t = [A^t | b^t]. i_Dt is not allowed to overlap with o_transform.
 * \param i_det Determimants (inverse) of i_Dt.
 * \param o_transform Extracted affine transformation (2x3) that is compatible with column-major
 * images and nppi -> [A^t | (-b2, -b1)^t] because of the identity Ax + b = A^t * x^t + (-b2, -b1).
 * o_transform is not allowed to overlap with i_Dt.
 * \param i_ld_patches The rows/ stride of i_Dt.
 * \param i_ld The stride between patches of i_Dt.
 * \param i_patch_size Size of the image patches.
 */
template<typename Float>
__global__ void least_squares_transforms_kernel(const Float* __restrict__ i_Dt,
												const Float* __restrict__ i_det,
												float* __restrict__ o_transform, uint i_ld_patches,
												uint i_ld, Size i_patch_size)
{
	// Skipping first column. This is the unit vector.
	const auto off_i1 = blockIdx.x * i_ld + i_ld_patches;
	const auto off_i2 = blockIdx.x * i_ld + 2 * i_ld_patches;
	const auto off_o = blockIdx.x * 3 * 3; // Size of projective transformations: 3x3.

	// The input transformation applies to the coordinate system scaled to [-1,1].
	// D^t = [A^t | b^t
	const auto d = i_det[blockIdx.x];
	const auto a_11 = static_cast<float>(d * i_Dt[off_i1 + 1]);
	const auto a_12 = static_cast<float>(d * i_Dt[off_i1 + 2]);
	const auto a_21 = static_cast<float>(d * i_Dt[off_i2 + 1]);
	const auto a_22 = static_cast<float>(d * i_Dt[off_i2 + 2]);
	const auto b_1 = static_cast<float>(d * i_Dt[off_i1]);
	const auto b_2 = static_cast<float>(d * i_Dt[off_i2]);

	const auto det = a_11 * a_22 - a_12 * a_21;

	// Only allow reasonable matrices (up to rounding errors).
	if(abs(det) > 0.01f && abs(det) < 10.0f)
	{
		perspective_matrix(a_11, a_12, b_1, a_21, a_22, b_2, i_patch_size, o_transform + off_o);
	}
	else // Identity
	{
		perspective_matrix(1.0f, 0.0f, 0.0f, 0.0f, 1.0f, 0.0f, i_patch_size, o_transform + off_o);
	}
}

//! Templated function that extracts the affine transformation from the least squares solution
//! of least_squares.
/*!
 * \param sol Solution from least_squares().
 * \param det Determinants (inverse) of the affine transformation.
 * \param size_patches Size of the image patches, corresponding to the transformations.
 * \param batch_no Determines the memory position of the stored transformations.
 * \return Perspective transformations.
 */
template<typename Float>
device_patches<float> least_squares_transforms(const device_patches<Float>& sol,
											   const thrust::device_vector<Float>& det,
											   const Size& size_patches)
{
	Expects(sol.patch_count() == det.size());

	const auto row_maj = false;
	auto transformations =
		device_patches<float>{patch_index{sol.patch_count()}, 3_rows, 3_cols, row_maj};

	const auto ld_patches = gsl::narrow_cast<uint>(sol.ld_patches());
	const auto ld = gsl::narrow_cast<uint>(sol.ld());
	const auto sol_ptr = sol.data().get();
	const auto det_ptr = det.data().get();
	auto trans_ptr = transformations.data().get();

	const auto blockDim = dim3{1};
	const auto gridDim = dim3{gsl::narrow_cast<uint>(sol.patch_count())};

	least_squares_transforms_kernel<<<gridDim, blockDim>>>(sol_ptr, det_ptr, trans_ptr, ld_patches,
														   ld, size_patches);
	check(hipDeviceSynchronize(), hipSuccess);

	return transformations;
}

//! Least squares for image moments.
/*!
 * \param lhs Image moments.
 * \param rhs Image moments.
 * \param patch_size Patch size associated to the affine transformations.
 * \param hipblas.h Initialized cublas handle.
 * \return Affine transformations associated to the image moments.
 */
template<typename Float>
device_patches<float> least_squares_solver_impl(const device_patches<Float>& lhs,
												const device_patches<Float>& rhs,
												const Size& patch_size, hipblasHandle_t hipblas.h)
{
	const auto solutions = least_squares(lhs, rhs, hipblas.h);
	const auto dets = determinant(lhs, rhs);

	return least_squares_transforms(solutions, dets, patch_size);
}

device_patches<float> least_squares_solver(const device_patches<float>& lhs,
										   const device_patches<float>& rhs, const Size& patch_size,
										   hipblasHandle_t handle)
{
	return least_squares_solver_impl(lhs, rhs, patch_size, handle);
}

device_patches<float> least_squares_solver(const device_patches<double>& lhs,
										   const device_patches<double>& rhs,
										   const Size& patch_size, hipblasHandle_t handle)
{
	return least_squares_solver_impl(lhs, rhs, patch_size, handle);
}