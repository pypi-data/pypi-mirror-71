#include "hip/hip_runtime.h"
#include "../include/device_matrix.h"
#include "../include/device_patches.h"
#include "../include/extract_patches.h"
#include "../include/vec3.h"

#include "../extern/gsl/gsl_assert"

#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>

//! A function that extracts all patches with full support from an image.
/*
 * \param i_image The input image.
 * \param i_patch_rows Row count of the extracted patches.
 * \param i_patch_cols Column count of the extracted patches.
 * \return Matrix with one patch per row. Patches are are stored in column-major format.
 */
template<typename T>
device_patches<T> extract_patches_impl(const device_matrix<T>& i_image, Rows i_patch_rows,
									   Cols i_patch_cols)
{
	Expects(i_image.col_maj()); // TODO: remove limitation
	Expects(i_patch_rows.value() <= i_image.rows());
	Expects(i_patch_cols.value() <= i_image.cols());

	const auto cols = gsl::narrow_cast<int>(i_image.cols()); // safe: small value
	const auto rows = gsl::narrow_cast<int>(i_image.rows()); // safe: small value
	const auto ld = i_image.ld();
	const auto p_rows = gsl::narrow_cast<int>(i_patch_rows.value()); // safe: small value
	const auto p_cols = gsl::narrow_cast<int>(i_patch_cols.value()); // safe: small value
	const auto sub_cols = cols - p_rows + 1; // row pixels with full support patches
	const auto sub_rows = rows - p_cols + 1; // col pixels with full support patches
	const auto p_count = sub_cols * sub_rows; // number of patches with full support
	const auto p_total = p_rows * p_cols; // elements of each patch
	const auto p_stride = sub_rows * p_total; // stride between sub_row times patches in the output
	const auto image_ptr = i_image.data().get();

	const auto p_col_maj = true;
	auto o_patches = device_patches<T>{patch_index{p_count}, i_patch_rows, i_patch_cols, p_col_maj};
	auto o_patches_ptr = o_patches.data().get();

	// Iterate over every pixel in the input image.
	const auto first = thrust::counting_iterator<std::size_t>{0};
	const auto last = first + i_image.total();

	// Each thread copies one pixel from the input image to the right position in the
	// corresponding patches.
	thrust::for_each(first, last, [=] __device__(std::size_t idx) {
		const auto row = static_cast<int>(idx % ld); // safe: small value
		const auto col = static_cast<int>(idx / ld); // safe: small value

		const auto row_begin = max(0, row - p_rows + 1);
		const auto row_end = min(sub_rows, row + 1);
		const auto col_begin = max(0, col - p_cols + 1);
		const auto col_end = min(sub_cols, col + 1);

		const auto val = image_ptr[idx];

		// pixel position in the current patch
		auto p_col_local = min(col, p_cols - 1); // max(0, col - sub_cols + 1);
		for(auto j = col_begin; j < col_end; j++)
		{
			// pixel position in the current patch
			auto p_row_local = min(row, p_rows - 1); // max(0, row - sub_rows + 1);
			for(auto i = row_begin; i < row_end; i++)
			{
				// Patches are stored continuously beginning with the first pixel with a full
				// supported patch and then following the memory layout of the input image.
				// The patches itself are stored continuously in column-major format.
				const auto p_begin = i * p_total + j * p_stride;
				const auto p_idx_local = p_row_local + p_col_local * p_rows;
				const auto p_idx = p_begin + p_idx_local;
				o_patches_ptr[p_idx] = val;

				p_row_local--;
			}

			p_col_local--;
		}
	});

	return o_patches;
}

device_patches<float> extract_patches(const device_matrix<float>& i_image, Rows i_patch_rows,
									  Cols i_patch_cols)
{
	return extract_patches_impl(i_image, i_patch_rows, i_patch_cols);
}

device_patches<float> extract_patches(const device_matrix<float>& i_image, const Size& i_patch_size)
{
	return extract_patches_impl(i_image, Rows{i_patch_size.rows()}, Cols{i_patch_size.cols()});
}

device_patches<double> extract_patches(const device_matrix<double>& i_image, Rows i_patch_rows,
									   Cols i_patch_cols)
{
	return extract_patches_impl(i_image, i_patch_rows, i_patch_cols);
}

device_patches<double> extract_patches(const device_matrix<double>& i_image,
									   const Size& i_patch_size)
{
	return extract_patches_impl(i_image, Rows{i_patch_size.rows()}, Cols{i_patch_size.cols()});
}

device_patches<vec3<float>> extract_patches(const device_matrix<vec3<float>>& i_image,
											Rows i_patch_rows, Cols i_patch_cols)
{
	return extract_patches_impl(i_image, i_patch_rows, i_patch_cols);
}

device_patches<vec3<float>> extract_patches(const device_matrix<vec3<float>>& i_image,
											const Size& i_patch_size)
{
	return extract_patches_impl(i_image, Rows{i_patch_size.rows()}, Cols{i_patch_size.cols()});
}

device_patches<vec3<double>> extract_patches(const device_matrix<vec3<double>>& i_image,
											 Rows i_patch_rows, Cols i_patch_cols)
{
	return extract_patches_impl(i_image, i_patch_rows, i_patch_cols);
}

device_patches<vec3<double>> extract_patches(const device_matrix<vec3<double>>& i_image,
											 const Size& i_patch_size)
{
	return extract_patches_impl(i_image, Rows{i_patch_size.rows()}, Cols{i_patch_size.cols()});
}