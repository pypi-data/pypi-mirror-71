#include "hip/hip_runtime.h"
#include "../include/device_matrix.h"
#include "../include/enums.h"
#include "../include/partition_NA.h"
#include "../include/quickselect.h"
#include "../include/rec_image.h"
#include "../include/utils.h"

#include "../extern/gsl/gsl_assert"

#include <hip/hip_runtime.h>
#include <>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/sequence.h>

//! A CUDA kernel that copies the projected patch pixels for each image pixel to a matrix.
/*!
 * BlockDim * GridDim >= i_cols_assigned * i_rows_assigned.
 * \param i_labels_idx The assigned label for each pixel. The size is
 * i_cols_assigned * i_rows_assigned.
 * \param i_cols_assigned The number of fully supported patches in column direction extracted from
 * the original image.
 * \param i_rows_assigned The number of fully supported patches in row direction extracted from
 * the original image.
 * \param i_labels The prototypical labels. Patches need to be stored continuously.
 * \param i_labels_cols Number of elements of one patch of i_labels.
 * \param i_labels_rows Number of labels in i_labels.
 * \param o_projection Contains the projected points for each image pixel. It is assumed that
 * i_labels_idx contains only the assigned labels for patches with a full support in the original
 * image. Thus this memory needs to be of size (i_cols_assigned + i_cols_patch - 1) *
 * (i_rows_assigned + i_rows_patch - 1) * max_proj_px. The output is row-major, where each row
 * contains the projected points for each pixel of the reconstructed image.
 */
template<typename T>
__global__ void copy_patches(const size_t* __restrict__ i_labels_idx, unsigned int i_cols_assigned,
							 unsigned int i_rows_assigned, const T* __restrict__ i_labels,
							 unsigned int i_labels_cols, unsigned int i_labels_rows,
							 T* __restrict__ o_projection)
{
	// Label index.
	const auto tid = threadIdx.x + blockIdx.x * blockDim.x;

	if(tid < i_cols_assigned * i_rows_assigned)
	{
		const auto i_off = i_labels_idx[tid] * i_labels_cols * i_labels_rows;
		const auto rows_img = i_rows_assigned + i_labels_rows - 1;

		// Pixel position in the original image that coincides with the upper left patch pixel.
		const auto px0_col = tid / i_rows_assigned;
		const auto px0_row = tid % i_rows_assigned;

		const auto max_proj_px =
			min(i_cols_assigned, i_labels_cols) * min(i_rows_assigned, i_labels_rows);

		// Copy the pixels of a fixed patch to the correct location of the projection.
		// Labels are stored column-major.
		auto label_px_idx = size_t{0};
		for(size_t j = 0; j < i_labels_cols; j++)
		{
			const auto px_col = px0_col + j;

			for(size_t i = 0; i < i_labels_rows; i++)
			{
				const auto px_row = px0_row + i;
				const auto img_px_idx = px_row + px_col * rows_img;
				const auto proj_row = img_px_idx * max_proj_px;
				const auto proj_idx = proj_row + (label_px_idx % max_proj_px);

				o_projection[proj_idx] = i_labels[i_off + label_px_idx];

				label_px_idx += 1;
			}
		}
	}
}

//! A function that collects in each row the pixel of every patch intersecting in the same pixel.
/*!
 * \param i_labels_idx The assigned label for each pixel.
 * \param i_labels The prototypical labels.
 * \return Matrix with projected patch values for one pixel in each row. Row size is (i_labels_idx +
 * size of labels). Row-major.
 */
template<typename T>
device_matrix<T> get_proj_points(const device_matrix<size_t>& i_labels_idx,
								 const device_patches<T>& i_labels)
{
	Expects(i_labels_idx.col_maj());
	Expects(i_labels.col_maj_patches());
	Expects(i_labels.patch_count() > 0);
	Expects(i_labels.total_per_patch() > 0);
	Expects(i_labels_idx.cols() > 0);
	Expects(i_labels_idx.rows() > 0);
	Expects(*thrust::max_element(i_labels_idx.cbegin(), i_labels_idx.cend()) <
			i_labels.patch_count());

	const auto cols_patch = gsl::narrow<unsigned int>(i_labels.cols_patches());
	const auto rows_patch = gsl::narrow<unsigned int>(i_labels.rows_patches());
	const auto cols_assigned = gsl::narrow<unsigned int>(i_labels_idx.cols());
	const auto rows_assigned = gsl::narrow<unsigned int>(i_labels_idx.rows());

	// This formula holds for any image and patch size!
	const auto cols_img = cols_assigned + cols_patch - 1;
	const auto rows_img = rows_assigned + rows_patch - 1;

	const auto max_proj_px =
		std::min(cols_assigned, cols_patch) * std::min(rows_assigned, rows_patch);
	const auto row_maj = false;
	auto proj =
		device_matrix<T>{Rows{cols_img * rows_img}, Cols{max_proj_px}, T{details::NA}, row_maj};

	const auto blockDim = dim3{1024};
	const auto gridDim = dim3{gsl::narrow<unsigned int>(i_labels_idx.total() / 1024 + 1)};

	copy_patches<<<gridDim, blockDim>>>(i_labels_idx.data().get(), cols_assigned, rows_assigned,
										i_labels.data().get(), cols_patch, rows_patch,
										proj.data().get());

	check(hipDeviceSynchronize(), hipSuccess);

	return proj;
}

template<typename T>
__device__ T mean(const T* ptr, size_t size)
{
	auto sum = T{0};
	for(size_t i = 0; i < size; i++)
	{
		sum += ptr[i];
	}

	return sum / static_cast<T>(size);
}

template<typename T>
__device__ vec3<T> mean(const vec3<T>* ptr, size_t size)
{
	auto sum = vec3<T>{0};
	for(size_t i = 0; i < size; i++)
	{
		sum += ptr[i];
	}

	return sum / static_cast<T>(size);
}

template<typename T>
device_matrix<T> rec_image_impl(const device_matrix<size_t>& i_labels_idx,
								const device_patches<T>& i_labels, rec_t i_type)
{
	if(i_type != rec_t::median && i_type != rec_t::mean)
	{
		std::runtime_error("Error in function rec_image. Reconstruction type is not supported.");
		return device_matrix<T>{Rows{}, Cols{}}; // Otherwise nvcc is complaining.
	}

	auto proj = get_proj_points(i_labels_idx, i_labels);
	Ensures(!proj.col_maj()); // continuous rows/ projected points

	const auto img_rows = i_labels_idx.rows() + i_labels.rows_patches() - 1;
	const auto img_cols = i_labels_idx.cols() + i_labels.cols_patches() - 1;
	auto img = device_matrix<T>{Rows{img_rows}, Cols{img_cols}, i_labels_idx.col_maj()};
	auto img_ptr = img.data().get();
	auto proj_ptr = proj.data().get();
	const auto proj_cols = gsl::narrow<int>(proj.cols());
	const auto row_stride = proj.ld();
	const auto begin = thrust::counting_iterator<size_t>{0};
	const auto end = begin + proj.rows();

	// Iterate over each row of the projection/ each pixel in the reconstructed image.
	if(i_type == rec_t::median)
	{
		thrust::for_each(begin, end, [=] __device__(size_t idx) {
			const auto offset = idx * row_stride;
			const auto n_size = partition_NA(proj_ptr + offset, proj_cols);

			img_ptr[idx] = quickselect(proj_ptr + offset, n_size, n_size / 2);
			if(n_size % 2 == 0)
			{
				img_ptr[idx] =
					0.5 * (img_ptr[idx] + quickselect(proj_ptr + offset, n_size, n_size / 2 - 1));
			}
		});
	}
	else if(i_type == rec_t::mean)
	{
		thrust::for_each(begin, end, [=] __device__(size_t idx) {
			const auto offset = idx * row_stride;
			const auto n_size = partition_NA(proj_ptr + offset, proj_cols);
			img_ptr[idx] = mean(proj_ptr + offset, n_size);
		});
	}
	else
	{
		throw std::runtime_error{"Error in rec_image_impl: Reconstruction type not supported."};
	}

	return img;
}

template<typename T>
device_matrix<T> rec_image_impl(const device_patches<T>& i_labels, const Size& i_img_size,
								rec_t i_type)
{
	const auto labels_idx = [i_img_size, &i_labels] {
		const auto labels_size = patch_count_size(i_img_size, i_labels.size_patches());

		Expects(i_labels.patch_count() == labels_size.total());

		return device_matrix<std::size_t>{d_seq(i_labels.patch_count()), Rows{labels_size.rows()},
										  Cols{labels_size.cols()}};
	}();

	return rec_image_impl(labels_idx, i_labels, i_type);
}

/*
 * template instantiations
 */

device_matrix<float> rec_image(const device_matrix<size_t>& i_labels_idx,
							   const device_patches<float>& i_labels, rec_t i_type)
{
	return rec_image_impl(i_labels_idx, i_labels, i_type);
}

device_matrix<double> rec_image(const device_matrix<size_t>& i_labels_idx,
								const device_patches<double>& i_labels, rec_t i_type)
{
	return rec_image_impl(i_labels_idx, i_labels, i_type);
}

device_matrix<vec3<float>> rec_image(const device_matrix<size_t>& i_labels_idx,
									 const device_patches<vec3<float>>& i_labels, rec_t i_type)
{
	return rec_image_impl(i_labels_idx, i_labels, i_type);
}

device_matrix<vec3<double>> rec_image(const device_matrix<size_t>& i_labels_idx,
									  const device_patches<vec3<double>>& i_labels, rec_t i_type)
{
	return rec_image_impl(i_labels_idx, i_labels, i_type);
}

device_matrix<float> rec_image(const device_patches<float>& i_labels, const Size& i_img_size,
							   rec_t i_type)
{
	return rec_image_impl(i_labels, i_img_size, i_type);
}

device_matrix<double> rec_image(const device_patches<double>& i_labels, const Size& i_img_size,
								rec_t i_type)
{
	return rec_image_impl(i_labels, i_img_size, i_type);
}

device_matrix<vec3<float>> rec_image(const device_patches<vec3<float>>& i_labels,
									 const Size& i_img_size, rec_t i_type)
{
	return rec_image_impl(i_labels, i_img_size, i_type);
}

device_matrix<vec3<double>> rec_image(const device_patches<vec3<double>>& i_labels,
									  const Size& i_img_size, rec_t i_type)
{
	return rec_image_impl(i_labels, i_img_size, i_type);
}